/*!!--------------------------------------------------------------------------!
 *!! libNEGF: a general library for Non-Equilibrium Greens functions.         !
 *!! Copyright (C) 2012 - 2026                                                !
 *!!                                                                          !
 *!! This file is part of libNEGF: a library for                              !
 *!! Non Equilibrium Green's Function calculation                             !
 *!!                                                                          !
 *!! Developers: Alessandro Pecchia, Daniele Soccodato                        !
 *!! Former Contributors: Gabriele Penazzi, Luca Latessa, Aldo Di Carlo       !
 *!!                                                                          !
 *!! libNEGF is free software: you can redistribute it and/or modify          !
 *!! it under the terms of the GNU Lesse General Public License as published  !
 *!! by the Free Software Foundation, either version 3 of the License, or     !
 *!! (at your option) any later version.                                      !
 *!!                                                                          !
 *!!  You should have received a copy of the GNU Lesser General Public        !
 *!!  License along with libNEGF.  If not, see                                !
 *!!  <http://www.gnu.org/licenses/>.                                         !
 *!!--------------------------------------------------------------------------!
 */

#include "hipblas.h"
#include "hipsolver.h"
#include <assert.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 1024
#define TILE_DIM 32
#define BLOCK_ROWS 8

__global__ void CaddKernel(
    hipComplex *c, const hipComplex alpha, const hipComplex *a,
    const hipComplex beta, const hipComplex *b, int size
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < size) {
        c[i].x = (alpha.x * a[i].x - alpha.y * a[i].y) +
                 (beta.x * b[i].x - beta.y * b[i].y);
        c[i].y = (alpha.x * a[i].y + alpha.y * a[i].x) +
                 (beta.x * b[i].y + beta.y * b[i].x);
    }
}

__global__ void ZaddKernel(
    hipDoubleComplex *c, const hipDoubleComplex alpha, const hipDoubleComplex *a,
    const hipDoubleComplex beta, const hipDoubleComplex *b, int size
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < size) {
        c[i].x = (alpha.x * a[i].x - alpha.y * a[i].y) +
                 (beta.x * b[i].x - beta.y * b[i].y);
        c[i].y = (alpha.x * a[i].y + alpha.y * a[i].x) +
                 (beta.x * b[i].y + beta.y * b[i].x);
    }
}

/*
__global__ void hermitian(cuComplex *odata, const cuComplex *idata)
{
  __shared__ cuComplex tile[TILE_DIM][TILE_DIM];

  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
  {
     odata[(y+j)*width + x].x = tile[threadIdx.x][threadIdx.y + j].x;
     odata[(y+j)*width + x].y = -tile[threadIdx.x][threadIdx.y + j].y;
  }
}
*/
__global__ void CinitKernel(hipComplex *a, int nrow) {
    int size;
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    size = nrow * nrow;
    if(i < size) {
        if(i % (nrow + 1) == 0) {
            a[i].x = 1.0;
            a[i].y = 0.0;
        } else {
            a[i].x = 0.0;
            a[i].y = 0.0;
        }
    }
}

__global__ void ZinitKernel(hipDoubleComplex *a, int nrow) {
    int size;
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    size = nrow * nrow;
    if(i < size) {
        if(i % (nrow + 1) == 0) {
            a[i].x = 1.0;
            a[i].y = 0.0;
        } else {
            a[i].x = 0.0;
            a[i].y = 0.0;
        }
    }
}

__global__ void DinitKernel(double *a, int nrow) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i < nrow) {
        a[i] = 1.0;
    }
}

__global__ void SinitKernel(float *a, int nrow) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i < nrow) {
        a[i] = 1.0;
    }
}

__global__ void CtraceKernel(
    hipComplex *a, int nrow, float *trace, bool *mask, int mask_present
) {
    int size;
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    size = nrow * nrow;
    if(mask_present == 0) {
        if(i < size) {
            if(i % (nrow + 1) == 0) {
                trace[i % nrow] = a[i].x;
            }
        }
    }
    if(mask_present == 1) {
        if(i < size) {
            if(i % (nrow + 1) == 0) {
                if(mask[i % nrow]) {
                    trace[i % nrow] = a[i].x;
                }
            }
        }
    }
}

__global__ void ZtraceKernel(
    hipDoubleComplex *a, int nrow, double *trace, bool *mask, int mask_present
) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int size = nrow * nrow;
    if(mask_present == 0) {
        if(i < size) {
            if(i % (nrow + 1) == 0) {
                trace[i % nrow] = a[i].x;
            }
        }
    }
    if(mask_present == 1) {
        if(i < size) {
            if(i % (nrow + 1) == 0) {
                if(mask[i % nrow]) {
                    trace[i % nrow] = a[i].x;
                }
            }
        }
    }
}

/*~-~-~-~-~-~-~-~-~-~-~-~-~-~ DATA MOVEMENT  ROUTINES
 * -~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-*/

extern "C" int cu_createMat(void **d_A, int bytecount) {
    hipError_t err;
    err = hipMalloc(d_A, bytecount);
    // printf("GPU Address: %p \n",*d_A);
    return err;
}

extern "C" int cu_copyMatH2D(void *h_A, void *d_A, int bytecount) {
    hipError_t err;
    // printf("copy %p to %p\n",h_A,d_A);
    err = hipMemcpy(d_A, h_A, bytecount, hipMemcpyHostToDevice);
    return err;
}

extern "C" int cu_copyMatD2H(void *h_A, void *d_A, int bytecount) {
    hipError_t err;
    // printf("copy %p to %p\n",d_A,h_A);
    err = hipMemcpy(h_A, d_A, bytecount, hipMemcpyDeviceToHost);
    return err;
}

extern "C" int cu_deleteMat(void *d_A) {
    hipError_t err;
    // printf("add_free: %p",d_A);
    err = hipFree(d_A);
    return err;
}

/*~-~-~-~-~-~-~-~-~-~-~-~-~-~ INIT/FINAL ROUTINES
 * -~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-*/

extern "C" int cu_cudaGetDeviceCount(int *count) {
    hipError_t err;

    err = hipGetDeviceCount(count);
    return err;
}

extern "C" int cu_cudaGetDeviceProperties(int device) {
    hipError_t err;
    hipDeviceProp_t prop;

    err = hipGetDeviceProperties(&prop, device);

    printf(" Found GPU: Device Name: %s\n", prop.name);
    printf(" TotalMemory: %lu\n", (unsigned long)prop.totalGlobalMem);
    printf(" Shared per block: %lu\n", (unsigned long)prop.sharedMemPerBlock);
    return err;
}

extern "C" int cu_cublasInit(hipblasHandle_t *hcublas) {
    hipblasStatus_t err;
    err = hipblasCreate(hcublas);
    if(err != 0) {
        printf("cublas create error: %d\n", err);
    }
    // printf("hcublas Addr: %p \n",*hcublas);
    return err;
}

extern "C" int cu_cublasFinalize(hipblasHandle_t hcublas) {
    hipblasStatus_t err;
    err = hipblasDestroy(hcublas);
    return err;
}

extern "C" int cu_cusolverInit(hipsolverHandle_t *hcusolver) {
    hipsolverStatus_t err;
    err = hipsolverDnCreate(hcusolver);
    if(err != 0) {
        printf("cusolver create error: %d\n", err);
    }
    // printf("hcusolver Addr: %p \n",*hcusolver);
    return err;
}

extern "C" int cu_cusolverFinalize(hipsolverHandle_t hcusolver) {
    hipsolverStatus_t err;
    err = hipsolverDnDestroy(hcusolver);
    return err;
}

/*~-~-~-~-~-~-~-~-~-~-~-~-~-~ MATRIX ROUTINES
 * -~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-*/
/* C = alpha op(A) op(B) + beta C
 * m: #rows of op(A)
 * n: #cols of op(B)
 * k: #cols of op(A) = #rows of op(B)
 */
extern "C" int cu_CmultMat(
    hipblasHandle_t hcublas, int m, int n, int k, hipComplex *alpha, void *d_A,
    void *d_B, hipComplex *beta, void *d_C, int dagger
) {
    hipComplex *pdA, *pdB, *pdC;

    // printf("A: %p B: %p C: %p\n",d_A,d_B,d_C);
    pdA = (hipComplex *)d_A;
    pdB = (hipComplex *)d_B;
    pdC = (hipComplex *)d_C;
    hipblasStatus_t err;
    if(dagger == 0) {
        err = hipblasCgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, pdA, m, pdB, k,
            beta, pdC, m
        );
    }
    if(dagger == 1) {
        err = hipblasCgemm(
            hcublas, HIPBLAS_OP_C, HIPBLAS_OP_N, m, n, k, alpha, pdA, k, pdB, k,
            beta, pdC, m
        );
    }
    if(dagger == 2) {
        err = hipblasCgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_C, m, n, k, alpha, pdA, m, pdB, n,
            beta, pdC, m
        );
    }
    return err;
}

//  C = alpha op(A) op(B) + beta C
// op(A):  m x k
// op(B):  k x n
//     C:  m x n
extern "C" int cu_ZmultMat(
    hipblasHandle_t hcublas, int m, int n, int k, hipDoubleComplex *alpha,
    void *d_A, void *d_B, hipDoubleComplex *beta, void *d_C, int dagger
) {
    hipDoubleComplex *pdA, *pdB, *pdC;

    pdA = (hipDoubleComplex *)d_A;
    pdB = (hipDoubleComplex *)d_B;
    pdC = (hipDoubleComplex *)d_C;
    hipblasStatus_t err;
    if(dagger == 0) {
        err = hipblasZgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, pdA, m, pdB, k,
            beta, pdC, m
        );
    }
    if(dagger == 1) {
        err = hipblasZgemm(
            hcublas, HIPBLAS_OP_C, HIPBLAS_OP_N, m, n, k, alpha, pdA, k, pdB, k,
            beta, pdC, m
        );
    }
    if(dagger == 2) {
        err = hipblasZgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_C, m, n, k, alpha, pdA, m, pdB, n,
            beta, pdC, m
        );
    }
    return err;
}

extern "C" int cu_Cinverse(
    hipblasHandle_t hcublas, hipsolverHandle_t hcusolver, void *d_A,
    void *d_Ainv, int N
) {
    hipError_t cudaStatus;
    hipsolverStatus_t cusolverStatus;
    hipblasStatus_t hipblasStatus_t;
    // declare arrays on the device
    hipComplex *pdA, *pdAinv, *d_LU, *d_Work;

    pdA = (hipComplex *)d_A;
    pdAinv = (hipComplex *)d_Ainv;
    // coeff . matrix , rhs , workspace
    int *d_pivot, *d_info, lwork; // pivots , info , worksp . size
    int info_gpu = 0;

    // compute buffer size and prep . memory
    cusolverStatus =
        hipsolverDnCgetrf_bufferSize(hcusolver, N, N, pdA, N, &lwork);
    // prepare memory on the device

    cudaStatus = hipMalloc((void **)&d_LU, N * N * sizeof(hipComplex));
    cudaStatus = hipMalloc((void **)&d_pivot, N * sizeof(int));
    cudaStatus = hipMalloc((void **)&d_info, sizeof(int));
    // copy d_LU <- pdA
    hipblasStatus_t = hipblasCcopy(hcublas, N * N, pdA, 1, d_LU, 1);

    cudaStatus = hipMalloc((void **)&d_Work, lwork * sizeof(hipComplex));

    // LU factorization of d_A , with partial pivoting and row
    // interchanges ; row i is interchanged with row d_pivot ( i );
    cusolverStatus =
        hipsolverDnCgetrf(hcusolver, N, N, d_LU, N, d_Work, d_pivot, d_info);

    // use the LU factorization to solve the system d_LU * x = d_Ainv ;
    // the solution overwrites d_Ainv
    cusolverStatus = hipsolverDnCgetrs(
        hcusolver, HIPBLAS_OP_N, N, N, d_LU, N, d_pivot, pdAinv, N, d_info
    );

    cudaStatus =
        hipMemcpy(&info_gpu, d_info, sizeof(int), hipMemcpyDeviceToHost);
    // d_info -> info_gpu
    cudaStatus = hipFree(d_pivot);
    cudaStatus = hipFree(d_info);
    cudaStatus = hipFree(d_Work);
    cudaStatus = hipFree(d_LU);
    return cudaStatus;
}

extern "C" int cu_Zinverse(
    hipblasHandle_t hcublas, hipsolverHandle_t hcusolver, void *d_A,
    void *d_Ainv, int N
) {
    hipError_t cudaStatus;
    hipsolverStatus_t cusolverStatus;
    hipblasStatus_t hipblasStatus_t;
    // declare arrays on the device
    hipDoubleComplex *pdA, *pdAinv, *d_LU, *d_Work;

    pdA = (hipDoubleComplex *)d_A;
    pdAinv = (hipDoubleComplex *)d_Ainv;
    // coeff . matrix , rhs , workspace
    int *d_pivot, *d_info, lwork; // pivots , info , worksp . size
    int info_gpu = 0;

    // compute buffer size and prep . memory
    cusolverStatus =
        hipsolverDnZgetrf_bufferSize(hcusolver, N, N, pdA, N, &lwork);
    // prepare memory on the device

    cudaStatus = hipMalloc((void **)&d_LU, N * N * sizeof(hipDoubleComplex));
    cudaStatus = hipMalloc((void **)&d_pivot, N * sizeof(int));
    cudaStatus = hipMalloc((void **)&d_info, sizeof(int));
    // copy d_LU <- pdA
    hipblasStatus_t = hipblasZcopy(hcublas, N * N, pdA, 1, d_LU, 1);

    cudaStatus = hipMalloc((void **)&d_Work, lwork * sizeof(hipDoubleComplex));

    // LU factorization of d_A , with partial pivoting and row
    // interchanges ; row i is interchanged with row d_pivot ( i );
    cusolverStatus =
        hipsolverDnZgetrf(hcusolver, N, N, d_LU, N, d_Work, d_pivot, d_info);

    // use the LU factorization to solve the system d_LU * x = d_Ainv ;
    // the solution overwrites d_Ainv
    cusolverStatus = hipsolverDnZgetrs(
        hcusolver, HIPBLAS_OP_N, N, N, d_LU, N, d_pivot, pdAinv, N, d_info
    );

    cudaStatus =
        hipMemcpy(&info_gpu, d_info, sizeof(int), hipMemcpyDeviceToHost);
    // d_info -> info_gpu
    cudaStatus = hipFree(d_pivot);
    cudaStatus = hipFree(d_info);
    cudaStatus = hipFree(d_Work);
    cudaStatus = hipFree(d_LU);
    return cudaStatus;
}

extern "C" int cu_Ckernelsum(
    void *d_C, hipComplex *alpha, void *d_A, hipComplex *beta, void *d_B, int size
) {
    int num_blocks;
    hipComplex *pdA = (hipComplex *)d_A;
    hipComplex *pdB = (hipComplex *)d_B;
    hipComplex *pdC = (hipComplex *)d_C;

    num_blocks = (size / BLOCK_SIZE) + 1;

    CaddKernel<<<num_blocks, BLOCK_SIZE>>>(pdC, *alpha, pdA, *beta, pdB, size);

    return 0;
}

extern "C" int cu_Zkernelsum(
    void *d_C, hipDoubleComplex *alpha, void *d_A, hipDoubleComplex *beta,
    void *d_B, int size
) {
    int num_blocks;
    hipDoubleComplex *pdA = (hipDoubleComplex *)d_A;
    hipDoubleComplex *pdB = (hipDoubleComplex *)d_B;
    hipDoubleComplex *pdC = (hipDoubleComplex *)d_C;

    num_blocks = (size / BLOCK_SIZE) + 1;

    ZaddKernel<<<num_blocks, BLOCK_SIZE>>>(pdC, *alpha, pdA, *beta, pdB, size);

    return 0;
}

extern "C" int cu_Cmatsum(
    hipblasHandle_t hcublas, int m, int n, hipComplex *alpha, void *d_A,
    hipComplex *beta, void *d_B, void *d_C, int dagger
) {
    // m number of rows of matrix op(A) and C
    // n number of columns of matrix op(B) and C
    hipComplex *pdA = (hipComplex *)d_A;
    hipComplex *pdB = (hipComplex *)d_B;
    hipComplex *pdC = (hipComplex *)d_C;

    hipblasStatus_t err;
    if(dagger == 0) {
        err = hipblasCgeam(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, alpha, pdA, m, beta, pdB,
            m, pdC, m
        );
    }
    if(dagger == 1) {
        err = hipblasCgeam(
            hcublas, HIPBLAS_OP_C, HIPBLAS_OP_N, m, n, alpha, pdA, n, beta, pdB,
            m, pdC, m
        );
    }
    if(dagger == 2) {
        err = hipblasCgeam(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_C, m, n, alpha, pdA, m, beta, pdB,
            n, pdC, m
        );
    }
    return err;
}

extern "C" int cu_Zmatsum(
    hipblasHandle_t hcublas, int m, int n, hipDoubleComplex *alpha, void *d_A,
    hipDoubleComplex *beta, void *d_B, void *d_C, int dagger
) {
    // m number of rows of matrix op(A) and C
    // n number of columns of matrix op(B) and C
    hipDoubleComplex *pdA = (hipDoubleComplex *)d_A;
    hipDoubleComplex *pdB = (hipDoubleComplex *)d_B;
    hipDoubleComplex *pdC = (hipDoubleComplex *)d_C;

    hipblasStatus_t err;
    if(dagger == 0) {
        err = hipblasZgeam(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, alpha, pdA, m, beta, pdB,
            m, pdC, m
        );
    }
    if(dagger == 1) {
        err = hipblasZgeam(
            hcublas, HIPBLAS_OP_C, HIPBLAS_OP_N, m, n, alpha, pdA, n, beta, pdB,
            m, pdC, m
        );
    }
    if(dagger == 2) {
        err = hipblasZgeam(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_C, m, n, alpha, pdA, m, beta, pdB,
            n, pdC, m
        );
    }
    return err;
}

extern "C" int cu_Cinitmat(void *d_A, int nrow) {
    int num_blocks;
    int size = nrow * nrow;
    hipComplex *pdA = (hipComplex *)d_A;

    num_blocks = (size / BLOCK_SIZE) + 1;

    CinitKernel<<<num_blocks, BLOCK_SIZE>>>(pdA, nrow);

    return 0;
}

extern "C" int cu_Zinitmat(void *d_A, int nrow) {
    int num_blocks;
    int size = nrow * nrow;
    hipDoubleComplex *pdA = (hipDoubleComplex *)d_A;

    num_blocks = (size / BLOCK_SIZE) + 1;

    ZinitKernel<<<num_blocks, BLOCK_SIZE>>>(pdA, nrow);

    return 0;
}

extern "C" float cu_Ctrace(
    hipblasHandle_t hcublas, void *d_A, int nrow, void *h_mask, int mask_present
) {
    hipComplex *pdA = (hipComplex *)d_A;
    int size = nrow * nrow;
    int num_blocks = (size / BLOCK_SIZE) + 1;
    float *d_work;
    hipError_t cudaStatus = hipMalloc((void **)&d_work, nrow * sizeof(float));
    float *d_iden;
    cudaStatus = hipMalloc((void **)&d_iden, nrow * sizeof(float));
    assert(cudaStatus == hipSuccess);
    bool *d_mask;
    cudaStatus = hipMalloc((void **)&d_mask, nrow * sizeof(bool));
    assert(cudaStatus == hipSuccess);
    if(h_mask) {
        cudaStatus = hipMemcpy(
            d_mask, h_mask, nrow * sizeof(bool), hipMemcpyHostToDevice
        );
        assert(cudaStatus == hipSuccess);
    }

    SinitKernel<<<num_blocks, BLOCK_SIZE>>>(d_iden, nrow);
    CtraceKernel<<<num_blocks, BLOCK_SIZE>>>(
        pdA, nrow, d_work, d_mask, mask_present
    );

    float result;
    hipblasStatus_t err =
        hipblasSdot(hcublas, nrow, d_iden, 1, d_work, 1, &result);
    assert(err == hipSuccess);

    cudaStatus = hipFree(d_work);
    cudaStatus = hipFree(d_iden);
    cudaStatus = hipFree(d_mask);

    return result;
}

extern "C" double cu_Ztrace(
    hipblasHandle_t hcublas, void *d_A, int nrow, void *h_mask, int mask_present
) {
    hipDoubleComplex *pdA = (hipDoubleComplex *)d_A;
    int size = nrow * nrow;
    int num_blocks = (size / BLOCK_SIZE) + 1;
    double *d_work;
    hipError_t cudaStatus =
        hipMalloc((void **)&d_work, nrow * sizeof(double));
    double *d_iden;
    cudaStatus = hipMalloc((void **)&d_iden, nrow * sizeof(double));
    assert(cudaStatus == hipSuccess);
    bool *d_mask;
    cudaStatus = hipMalloc((void **)&d_mask, nrow * sizeof(bool));
    assert(cudaStatus == hipSuccess);
    if(h_mask) {
        cudaStatus = hipMemcpy(
            d_mask, h_mask, nrow * sizeof(bool), hipMemcpyHostToDevice
        );
        assert(cudaStatus == hipSuccess);
    }

    DinitKernel<<<num_blocks, BLOCK_SIZE>>>(d_iden, nrow);
    ZtraceKernel<<<num_blocks, BLOCK_SIZE>>>(
        pdA, nrow, d_work, d_mask, mask_present
    );

    double result;
    hipblasStatus_t err =
        hipblasDdot(hcublas, nrow, d_iden, 1, d_work, 1, &result);
    assert(err == hipSuccess);

    cudaStatus = hipFree(d_work);
    cudaStatus = hipFree(d_iden);
    cudaStatus = hipFree(d_mask);

    return result;
}

extern "C" int
cu_Cmatcopy(hipblasHandle_t hcublas, void *d_A, void *d_B, int N) {
    hipComplex *pdA = (hipComplex *)d_A;
    hipComplex *pdB = (hipComplex *)d_B;

    hipblasStatus_t err;

    err = hipblasCcopy(hcublas, N * N, pdA, 1, pdB, 1);
    return err;
}

extern "C" int
cu_Zmatcopy(hipblasHandle_t hcublas, void *d_A, void *d_B, int size) {
    hipDoubleComplex *pdA = (hipDoubleComplex *)d_A;
    hipDoubleComplex *pdB = (hipDoubleComplex *)d_B;

    hipblasStatus_t err;

    err = hipblasZcopy(hcublas, size, pdA, 1, pdB, 1);
    return err;
}

extern "C" int cu_Casum(hipblasHandle_t hcublas, void *d_A, float *summ, int N) {
    hipComplex *pdA = (hipComplex *)d_A;

    hipblasStatus_t err;

    err = hipblasScasum(hcublas, N, pdA, 1, summ);
    return err;
}

extern "C" int
cu_Zasum(hipblasHandle_t hcublas, void *d_A, double *summ, int N) {
    hipDoubleComplex *pdA = (hipDoubleComplex *)d_A;

    hipblasStatus_t err;

    err = hipblasDzasum(hcublas, N, pdA, 1, summ);
    return err;
}

extern "C" int cu_Cdecimation(
    hipblasHandle_t hcublas, hipsolverHandle_t hcusolver, void *h_Go_out,
    void *h_Ao_in, void *h_Bo_in, void *h_Co_in, int n, int tf32, int *ncyc,
    hipComplex *one, hipComplex *mone, hipComplex *zero, float SGFACC
) {
    hipError_t cudaStatus;
    hipsolverStatus_t cusolverStatus;
    hipblasStatus_t hipblasStatus_t;

    float summ;
    bool okCo = false;
    hipComplex *phGo_out = (hipComplex *)h_Go_out;
    hipComplex *phAo_in = (hipComplex *)h_Ao_in;
    hipComplex *phBo_in = (hipComplex *)h_Bo_in;
    hipComplex *phCo_in = (hipComplex *)h_Co_in;

    hipComplex *d_Ao, *d_Bo, *d_Co, *d_Go, *d_Ao_s, *d_C1, *d_T, *d_Self,
        *d_work;
    int *d_pivot, *d_info, lwork; // pivots , info , worksp . size
    int i1, num_blocks;

    num_blocks = ((n * n) / BLOCK_SIZE) + 1;

    cudaStatus = hipMalloc((void **)&d_Ao, n * n * sizeof(hipComplex));
    cudaStatus = hipMalloc((void **)&d_Bo, n * n * sizeof(hipComplex));
    cudaStatus = hipMalloc((void **)&d_Co, n * n * sizeof(hipComplex));
    cudaStatus = hipMemcpy(
        d_Ao, phAo_in, n * n * sizeof(hipComplex), hipMemcpyHostToDevice
    );
    cudaStatus = hipMemcpy(
        d_Bo, phBo_in, n * n * sizeof(hipComplex), hipMemcpyHostToDevice
    );
    cudaStatus = hipMemcpy(
        d_Co, phCo_in, n * n * sizeof(hipComplex), hipMemcpyHostToDevice
    );

    hipblasStatus_t = hipblasSetPointerMode(hcublas, HIPBLAS_POINTER_MODE_HOST);
    if(tf32 == 1) {
        hipblasStatus_t = hipblasSetMathMode(hcublas, HIPBLAS_TENSOR_OP_MATH);
    }

    cudaStatus = hipMalloc((void **)&d_Ao_s, n * n * sizeof(hipComplex));
    cudaStatus = hipMalloc((void **)&d_C1, n * n * sizeof(hipComplex));
    cudaStatus = hipMalloc((void **)&d_Go, n * n * sizeof(hipComplex));
    cudaStatus = hipMalloc((void **)&d_pivot, n * sizeof(int));
    cudaStatus = hipMalloc((void **)&d_T, n * n * sizeof(hipComplex));
    cudaStatus = hipMalloc((void **)&d_Self, n * n * sizeof(hipComplex));
    cudaStatus = hipMalloc((void **)&d_info, sizeof(int));

    cusolverStatus =
        hipsolverDnCgetrf_bufferSize(hcusolver, n, n, d_Self, n, &lwork);
    cudaStatus = hipMalloc((void **)&d_work, lwork * sizeof(hipComplex));

    hipblasStatus_t = hipblasCcopy(hcublas, n * n, d_Ao, 1, d_Ao_s, 1);

    for(i1 = 1; i1 <= 300; i1++) {
        *ncyc = i1;

        CinitKernel<<<num_blocks, BLOCK_SIZE>>>(d_Go, n);

        hipblasStatus_t = hipblasCcopy(hcublas, n * n, d_Ao, 1, d_Self, 1);

        cusolverStatus = hipsolverDnCgetrf(
            hcusolver, n, n, d_Self, n, d_work, d_pivot, d_info
        );
        cusolverStatus = hipsolverDnCgetrs(
            hcusolver, HIPBLAS_OP_N, n, n, d_Self, n, d_pivot, d_Go, n, d_info
        );

        hipblasStatus_t = hipblasCgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Go, n, d_Co, n,
            zero, d_T, n
        );

        hipblasStatus_t = hipblasCgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Co, n, d_T, n,
            zero, d_C1, n
        );

        hipblasStatus_t = hipblasScasum(hcublas, n * n, d_C1, 1, &summ);
        // printf("loop it= %d , summ= %f \n ", i1, summ);

        if(summ <= SGFACC) {
            if(okCo) {
                break;
            } else {
                okCo = true;
            }
        } else {
            okCo = false;
        }

        hipblasStatus_t = hipblasCgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Bo, n, d_T, n,
            zero, d_Self, n
        );

        hipblasStatus_t = hipblasCaxpy(hcublas, n * n, mone, d_Self, 1, d_Ao_s, 1);
        hipblasStatus_t = hipblasCaxpy(hcublas, n * n, mone, d_Self, 1, d_Ao, 1);

        hipblasStatus_t = hipblasCgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Go, n, d_Bo, n,
            zero, d_T, n
        );
        hipblasStatus_t = hipblasCgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, mone, d_Co, n, d_T, n,
            one, d_Ao, n
        );

        hipblasStatus_t = hipblasCcopy(hcublas, n * n, d_C1, 1, d_Co, 1);

        hipblasStatus_t = hipblasCgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Bo, n, d_T, n,
            zero, d_C1, n
        );

        hipblasStatus_t = hipblasCcopy(hcublas, n * n, d_C1, 1, d_Bo, 1);
    }

    CinitKernel<<<num_blocks, BLOCK_SIZE>>>(d_Go, n);
    hipblasStatus_t = hipblasCcopy(hcublas, n * n, d_Ao_s, 1, d_Self, 1);
    cusolverStatus =
        hipsolverDnCgetrf(hcusolver, n, n, d_Self, n, d_work, d_pivot, d_info);
    cusolverStatus = hipsolverDnCgetrs(
        hcusolver, HIPBLAS_OP_N, n, n, d_Self, n, d_pivot, d_Go, n, d_info
    );

    // cublasStatus = cublasCcopy(hcublas, n*n, d_Go, 1, d_Go_out, 1);

    cudaStatus = hipMemcpy(
        phGo_out, d_Go, n * n * sizeof(hipComplex), hipMemcpyDeviceToHost
    );

    cudaStatus = hipFree(d_Ao);
    cudaStatus = hipFree(d_Bo);
    cudaStatus = hipFree(d_Co);
    cudaStatus = hipFree(d_Go);
    cudaStatus = hipFree(d_Ao_s);
    cudaStatus = hipFree(d_C1);
    cudaStatus = hipFree(d_pivot);
    cudaStatus = hipFree(d_T);
    cudaStatus = hipFree(d_Self);
    cudaStatus = hipFree(d_info);
    cudaStatus = hipFree(d_work);

    return cudaStatus;
}

extern "C" int cu_Zdecimation(
    hipblasHandle_t hcublas, hipsolverHandle_t hcusolver, void *h_Go_out,
    void *h_Ao_in, void *h_Bo_in, void *h_Co_in, int n, int tf32, int *ncyc,
    hipDoubleComplex *one, hipDoubleComplex *mone, hipDoubleComplex *zero,
    double SGFACC
) {
    assert(h_Go_out);
    assert(h_Ao_in);
    assert(h_Bo_in);
    assert(h_Co_in);
    assert(n > 0);
    assert(tf32 == 0 || tf32 == 1);
    assert(ncyc);
    assert(one);
    assert(mone);
    assert(zero);
    assert(SGFACC > 0.0);

    int num_elements = n * n;
    int num_blocks = (num_elements / BLOCK_SIZE) + 1;

    hipDoubleComplex *d_Ao;
    hipError_t cudaStatus =
        hipMalloc((void **)&d_Ao, num_elements * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);

    hipDoubleComplex *d_Bo;
    cudaStatus =
        hipMalloc((void **)&d_Bo, num_elements * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);

    hipDoubleComplex *d_Co;
    cudaStatus =
        hipMalloc((void **)&d_Co, num_elements * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);

    cudaStatus = hipMemcpy(
        d_Ao, h_Ao_in, n * n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice
    );
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipMemcpy(
        d_Bo, h_Bo_in, n * n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice
    );
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipMemcpy(
        d_Co, h_Co_in, n * n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice
    );
    assert(cudaStatus == hipSuccess);

    hipblasStatus_t hipblasStatus_t =
        hipblasSetPointerMode(hcublas, HIPBLAS_POINTER_MODE_HOST);
    assert(hipblasStatus_t == hipSuccess);

    if(tf32 == 1) {
        hipblasStatus_t = hipblasSetMathMode(hcublas, HIPBLAS_TENSOR_OP_MATH);
        assert(hipblasStatus_t == hipSuccess);
    }

    hipDoubleComplex *d_Ao_s;
    cudaStatus = hipMalloc((void **)&d_Ao_s, n * n * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);
    hipDoubleComplex *d_C1;
    cudaStatus = hipMalloc((void **)&d_C1, n * n * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);
    hipDoubleComplex *d_Go;
    cudaStatus = hipMalloc((void **)&d_Go, n * n * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);
    int *d_pivot;
    cudaStatus = hipMalloc((void **)&d_pivot, n * sizeof(int));
    assert(cudaStatus == hipSuccess);
    hipDoubleComplex *d_T;
    cudaStatus = hipMalloc((void **)&d_T, n * n * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);
    hipDoubleComplex *d_Self;
    cudaStatus = hipMalloc((void **)&d_Self, n * n * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);
    int *d_info;
    cudaStatus = hipMalloc((void **)&d_info, sizeof(int));
    assert(cudaStatus == hipSuccess);

    int lwork;
    hipsolverStatus_t cusolverStatus =
        hipsolverDnZgetrf_bufferSize(hcusolver, n, n, d_Self, n, &lwork);
    assert(cusolverStatus == hipSuccess);
    hipDoubleComplex *d_work;
    cudaStatus = hipMalloc((void **)&d_work, lwork * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);

    hipblasStatus_t = hipblasZcopy(hcublas, n * n, d_Ao, 1, d_Ao_s, 1);
    assert(hipblasStatus_t == hipSuccess);

    bool okCo = false;
    for(int i1 = 1; i1 <= 300; i1++) {
        *ncyc = i1;

        ZinitKernel<<<num_blocks, BLOCK_SIZE>>>(d_Go, n);

        hipblasStatus_t = hipblasZcopy(hcublas, n * n, d_Ao, 1, d_Self, 1);
        assert(hipblasStatus_t == hipSuccess);

        cusolverStatus = hipsolverDnZgetrf(
            hcusolver, n, n, d_Self, n, d_work, d_pivot, d_info
        );
        assert(cusolverStatus == hipSuccess);
        cusolverStatus = hipsolverDnZgetrs(
            hcusolver, HIPBLAS_OP_N, n, n, d_Self, n, d_pivot, d_Go, n, d_info
        );
        assert(cusolverStatus == hipSuccess);

        hipblasStatus_t = hipblasZgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Go, n, d_Co, n,
            zero, d_T, n
        );
        assert(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = hipblasZgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Co, n, d_T, n,
            zero, d_C1, n
        );
        assert(hipblasStatus_t == hipSuccess);

        double summ;
        hipblasStatus_t = hipblasDzasum(hcublas, n * n, d_C1, 1, &summ);
        assert(hipblasStatus_t == hipSuccess);
        // printf("loop it= %d , summ= %f \n ", i1, summ);

        if(summ <= SGFACC) {
            if(okCo) {
                break;
            } else {
                okCo = true;
            }
        } else {
            okCo = false;
        }

        hipblasStatus_t = hipblasZgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Bo, n, d_T, n,
            zero, d_Self, n
        );
        assert(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = hipblasZaxpy(hcublas, n * n, mone, d_Self, 1, d_Ao_s, 1);
        assert(hipblasStatus_t == hipSuccess);
        hipblasStatus_t = hipblasZaxpy(hcublas, n * n, mone, d_Self, 1, d_Ao, 1);
        assert(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = hipblasZgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Go, n, d_Bo, n,
            zero, d_T, n
        );
        assert(hipblasStatus_t == hipSuccess);
        hipblasStatus_t = hipblasZgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, mone, d_Co, n, d_T, n,
            one, d_Ao, n
        );
        assert(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = hipblasZcopy(hcublas, n * n, d_C1, 1, d_Co, 1);
        assert(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = hipblasZgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Bo, n, d_T, n,
            zero, d_C1, n
        );
        assert(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = hipblasZcopy(hcublas, n * n, d_C1, 1, d_Bo, 1);
        assert(hipblasStatus_t == hipSuccess);
    }

    ZinitKernel<<<num_blocks, BLOCK_SIZE>>>(d_Go, n);
    hipblasStatus_t = hipblasZcopy(hcublas, n * n, d_Ao_s, 1, d_Self, 1);
    assert(hipblasStatus_t == hipSuccess);
    cusolverStatus =
        hipsolverDnZgetrf(hcusolver, n, n, d_Self, n, d_work, d_pivot, d_info);
    assert(cusolverStatus == hipSuccess);
    cusolverStatus = hipsolverDnZgetrs(
        hcusolver, HIPBLAS_OP_N, n, n, d_Self, n, d_pivot, d_Go, n, d_info
    );
    assert(cusolverStatus == hipSuccess);

    cudaStatus = hipMemcpy(
        h_Go_out, d_Go, n * n * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost
    );

    cudaStatus = hipFree(d_pivot);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_info);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Ao);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Bo);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Co);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Go);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Ao_s);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_C1);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_T);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Self);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_work);
    assert(cudaStatus == hipSuccess);

    return cudaStatus;
}
