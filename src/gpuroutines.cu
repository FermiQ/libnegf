#include "hip/hip_runtime.h"
/*!!--------------------------------------------------------------------------!
 *!! libNEGF: a general library for Non-Equilibrium Greens functions.         !
 *!! Copyright (C) 2012 - 2026                                                !
 *!!                                                                          !
 *!! This file is part of libNEGF: a library for                              !
 *!! Non Equilibrium Green's Function calculation                             !
 *!!                                                                          !
 *!! Developers: Alessandro Pecchia, Daniele Soccodato                        !
 *!! Former Contributors: Gabriele Penazzi, Luca Latessa, Aldo Di Carlo       !
 *!!                                                                          !
 *!! libNEGF is free software: you can redistribute it and/or modify          !
 *!! it under the terms of the GNU Lesse General Public License as published  !
 *!! by the Free Software Foundation, either version 3 of the License, or     !
 *!! (at your option) any later version.                                      !
 *!!                                                                          !
 *!!  You should have received a copy of the GNU Lesser General Public        !
 *!!  License along with libNEGF.  If not, see                                !
 *!!  <http://www.gnu.org/licenses/>.                                         !
 *!!--------------------------------------------------------------------------!
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <cassert>
#include <climits>
#include <cstddef>
#include <cstdint>
#include <cstdio>
#include <cstdlib>

#include <type_traits>

#include <libnegf/hipblas.h>
#include <libnegf/hipsolver.h>


#if __CUDA_ARCH__
#define ENFORCE(expr) \
    do { \
        if(!(expr)) { \
            printf( \
                "%s:%u: ENFORCEMENT FAILURE: %s is FALSE\n", __FILE__, \
                __LINE__, #expr \
            ); \
            __trap(); \
        } \
    } while(false)
#else
#define ENFORCE(expr) \
    do { \
        if(!(expr)) { \
            std::fprintf( \
                stderr, "%s:%u: ENFORCEMENT FAILURE: %s is FALSE\n", __FILE__, \
                __LINE__, #expr \
            ); \
            std::exit(EXIT_FAILURE); \
        } \
    } while(false)
#endif


constexpr auto BLOCK_SIZE = std::size_t{1024};


/**
 * Given a floating-point type, returns the associated real-valued type; for
 * real-valued types, this is the type itself.
 */
template<typename>
struct get_real {};

template<>
struct get_real<float> {
    using type = float;
};

template<>
struct get_real<double> {
    using type = double;
};

template<>
struct get_real<hipComplex> {
    using type = float;
};

template<>
struct get_real<hipDoubleComplex> {
    using type = double;
};


/**
 * Computes c = α a · β b.
 */
template<typename Number>
__global__ void addKernel(
    Number* c, Number alpha, const Number* a, Number beta, const Number* b,
    size_t size
) {
    ENFORCE(c);
    ENFORCE(a);
    ENFORCE(b);

    auto i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < size) {
        c[i].x = (alpha.x * a[i].x - alpha.y * a[i].y) +
                 (beta.x * b[i].x - beta.y * b[i].y);
        c[i].y = (alpha.x * a[i].y + alpha.y * a[i].x) +
                 (beta.x * b[i].y + beta.y * b[i].x);
    }
}


/*
__global__ void hermitian(hipComplex *odata, const hipComplex *idata)
{
  __shared__ hipComplex tile[TILE_DIM][TILE_DIM];

  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
  {
     odata[(y+j)*width + x].x = tile[threadIdx.x][threadIdx.y + j].x;
     odata[(y+j)*width + x].y = -tile[threadIdx.x][threadIdx.y + j].y;
  }
}
*/


/**
 * Initializes a square complex matrix as the identity matrix.
 */
template<typename Number>
__global__ void initKernel(Number* a, size_t nrow) {
    using Real = typename get_real<Number>::type;

    ENFORCE(a);

    auto size = nrow * nrow;
    auto i = blockDim.x * blockIdx.x + threadIdx.x;
    auto one = Number{Real{1}};
    auto zero = Number{Real{0}};

    if(i < size) {
        if(i % (nrow + 1) == 0) {
            a[i] = one;
        } else {
            a[i] = zero;
        }
    }
}


template<typename Number>
__global__ void initArrayWithOnes(Number* a, size_t nrow) {
    static_assert(std::is_same_v<Number, typename get_real<Number>::type>);

    assert(a);

    auto i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i < nrow) {
        a[i] = Number{1};
    }
}


/**
 * Computes the trace of a matrix A.
 *
 * Optionally, a bit mask can be passed to the function. If a mask is present,
 * then a diagonal element (i,i) is only considered for the trace computation
 * if the i-th value in mask is nonzero.
 *
 * @param[in] mask A bitmask of length nrow indicating which rows to ignore
 * (zero means ignore).
 */
template<typename Number, typename Real = typename get_real<Number>::type>
__global__ void
traceKernel(Number* a, size_t nrow, Real* trace, bool* mask, int mask_present) {
    ENFORCE(a);
    ENFORCE(trace);
    ENFORCE(mask || mask_present == 0);
    ENFORCE(mask_present == 0 || mask_present == 1);

    auto size = nrow * nrow;
    auto i = blockDim.x * blockIdx.x + threadIdx.x;

    if(mask_present == 0) {
        if(i < size) {
            if(i % (nrow + 1) == 0) {
                trace[i % nrow] = a[i].x;
            }
        }
    }
    if(mask_present == 1) {
        if(i < size) {
            if(i % (nrow + 1) == 0) {
                if(mask[i % nrow]) {
                    trace[i % nrow] = a[i].x;
                } else {
                    trace[i % nrow] = 0.0;
                }
            }
        }
    }
}


/*
 * DATA MOVEMENT ROUTINES
 */

extern "C" int cu_createMat(void** d_A, size_t bytecount) {
    ENFORCE(d_A);
    hipError_t err = hipMalloc(d_A, bytecount);
    //printf("create mat at GPU Address: %p \n",*d_A);
    return err;
}

extern "C" int cu_cudaFreeAsync(void** d_A) {
    if(!*d_A) {
        return 0;
    }
    int stat = hipFreeAsync(*d_A, 0);
    *d_A = NULL;
    return stat;
}

extern "C" int cu_cudaMallocAsync(void** d_A, size_t bytecount) {
    ENFORCE(d_A);
    hipError_t err = hipMallocAsync(d_A, bytecount, 0);
    //printf("create mat at GPU Address: %p \n",*d_A);
    ENFORCE(err == hipSuccess);
    return err;
}

extern "C" int cu_copyMatH2D(void* h_A, void* d_A, size_t bytecount) {
    ENFORCE(h_A);
    ENFORCE(d_A);
    // printf("copy %p to %p\n",h_A,d_A);
    hipError_t err = hipMemcpy(d_A, h_A, bytecount, hipMemcpyHostToDevice);
    return err;
}

extern "C" int cu_copyMatH2D_async(void* h_A, void* d_A, size_t bytecount) {
    ENFORCE(h_A);
    ENFORCE(d_A);
    // printf("copy %p to %p\n",h_A,d_A);
    hipError_t err =
        hipMemcpyAsync(d_A, h_A, bytecount, hipMemcpyHostToDevice);
    return err;
}

extern "C" int cu_copyMatD2H(void* h_A, void* d_A, size_t bytecount) {
    ENFORCE(h_A);
    ENFORCE(d_A);

    hipError_t err = hipMemcpy(h_A, d_A, bytecount, hipMemcpyDeviceToHost);
    return err;
}

extern "C" int cu_deleteMat(void** d_A) {
    int stat = 0;
    if(*d_A != NULL) {
        stat = hipFree(*d_A);
        *d_A = NULL;
    }
    return stat;
}


/*
 * INIT/FINAL ROUTINES
 */

extern "C" int cu_cudaGetDeviceCount(int* count) {
    ENFORCE(count);
    hipError_t err = hipGetDeviceCount(count);
    ENFORCE(err == hipSuccess);
    return err;
}

extern "C" int cu_cudaGetDeviceProperties(int device) {
    hipDeviceProp_t prop;
    hipError_t err = hipGetDeviceProperties(&prop, device);
    ENFORCE(err == hipSuccess);

    printf(" Found GPU: Device Name: %s\n", prop.name);
    printf(" TotalMemory: %lu\n", (unsigned long)prop.totalGlobalMem);
    printf(" Shared per block: %lu\n", (unsigned long)prop.sharedMemPerBlock);

    return err;
}

extern "C" int cu_cudaSetDevice(int count) {
    hipError_t err = hipSetDevice(count);
    ENFORCE(err == hipSuccess);
    return err;
}

extern "C" int cu_cublasInit(hipblasHandle_t* hcublas) {
    ENFORCE(hcublas);
    hipblasStatus_t err = hipblasCreate(hcublas);
    ENFORCE(err == HIPBLAS_STATUS_SUCCESS);
    if(err != HIPBLAS_STATUS_SUCCESS) {
        printf("cublas create error: %d\n", err);
    }
    // printf("hcublas Addr: %p \n",*hcublas);
    return err;
}

extern "C" int cu_cublasFinalize(hipblasHandle_t hcublas) {
    hipblasStatus_t err = hipblasDestroy(hcublas);
    ENFORCE(err == HIPBLAS_STATUS_SUCCESS);
    return err;
}

extern "C" int cu_cusolverInit(hipsolverHandle_t* hcusolver) {
    ENFORCE(hcusolver);
    hipsolverStatus_t err = hipsolverDnCreate(hcusolver);
    ENFORCE(err == hipSuccess);
    if(err != 0) {
        printf("cusolver create error: %d\n", err);
    }
    // printf("hcusolver Addr: %p \n",*hcusolver);
    return err;
}

extern "C" int cu_cusolverFinalize(hipsolverHandle_t hcusolver) {
    hipsolverStatus_t err = hipsolverDnDestroy(hcusolver);
    ENFORCE(err == hipSuccess);
    return err;
}


/*
 * MATRIX ROUTINES
 */

/**
 * Multiplies the matrices A, B and adds the result to C.
 *
 * op(.) indicates if the matrix or its complex-conjugate is used. The allowed
 * values for dagger are:
 * * `dagger == 0`: compute C ≔ α A · B + β C
 * * `dagger == 1`: compute C ≔ α A^* · B + β C
 * * `dagger == 2`: compute C ≔ α A · B^* + β C
 *
 * @param[in] m The number of rows of C and op(A).
 * @param[in] n The number of columns of C and op(B).
 * @param[in] k The number of columns of op(A) and the number of rows of op(B).
 * @param[in] dagger A shorthand for various combinations of op(A), op(B).
 */
template<typename Number>
int cu_multMat(
    hipblasHandle_t hcublas, size_t m, size_t n, size_t k, const Number* alpha,
    const Number* d_A, const Number* d_B, const Number* beta, Number* d_C,
    int dagger
) {
    ENFORCE(alpha);
    ENFORCE(d_A);
    ENFORCE(d_B);
    ENFORCE(beta);
    ENFORCE(dagger == 0 || dagger == 1 || dagger == 2);

    hipblasStatus_t err;
    if(dagger == 0) {
        err = libnegf::cublasGemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, d_A, m, d_B, k,
            beta, d_C, m
        );
    } else if(dagger == 1) {
        err = libnegf::cublasGemm(
            hcublas, HIPBLAS_OP_C, HIPBLAS_OP_N, m, n, k, alpha, d_A, k, d_B, k,
            beta, d_C, m
        );
    } else if(dagger == 2) {
        err = libnegf::cublasGemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_C, m, n, k, alpha, d_A, m, d_B, n,
            beta, d_C, m
        );
    } else {
        std::fprintf(stderr, "expected dagger in [0, 1, 2], got %d\n", dagger);
        std::exit(EXIT_FAILURE);
    }
    ENFORCE(err == HIPBLAS_STATUS_SUCCESS);
    return err;
}

extern "C" int cu_CmultMat(
    hipblasHandle_t hcublas, size_t m, size_t n, size_t k,
    const hipComplex* alpha, const hipComplex* d_A, const hipComplex* d_B,
    const hipComplex* beta, hipComplex* d_C, int dagger
) {
    return cu_multMat(hcublas, m, n, k, alpha, d_A, d_B, beta, d_C, dagger);
}

extern "C" int cu_ZmultMat(
    hipblasHandle_t hcublas, size_t m, size_t n, size_t k,
    const hipDoubleComplex* alpha, const hipDoubleComplex* d_A,
    const hipDoubleComplex* d_B, const hipDoubleComplex* beta,
    hipDoubleComplex* d_C, int dagger
) {
    return cu_multMat(hcublas, m, n, k, alpha, d_A, d_B, beta, d_C, dagger);
}


template<typename Number>
int inverse(
    hipblasHandle_t hcublas, hipsolverHandle_t hcusolver, Number* d_A,
    Number* d_Ainv, size_t n
) {
    ENFORCE(hcusolver);
    ENFORCE(d_A);
    ENFORCE(d_Ainv);

    // compute buffer size and prep . memory
    int lwork;
    hipsolverStatus_t cusolverStatus =
        libnegf::cusolverDngetrf_bufferSize(hcusolver, n, n, d_A, n, &lwork);
    ENFORCE(cusolverStatus == HIPSOLVER_STATUS_SUCCESS);

    // prepare memory on the device
    Number* d_LU;
    hipError_t cudaStatus = hipMalloc((void**)&d_LU, n * n * sizeof(Number));
    ENFORCE(cudaStatus == hipSuccess);

    int* d_pivot;
    cudaStatus = hipMalloc((void**)&d_pivot, n * sizeof(int));
    ENFORCE(cudaStatus == hipSuccess);
    int* d_info;
    cudaStatus = hipMalloc((void**)&d_info, sizeof(int));
    ENFORCE(cudaStatus == hipSuccess);
    // copy d_LU <- pdA
    auto hipblasStatus_t = libnegf::cublasCopy(hcublas, n * n, d_A, 1, d_LU, 1);
    ENFORCE(hipblasStatus_t == HIPBLAS_STATUS_SUCCESS);

    Number* d_work;
    cudaStatus = hipMalloc((void**)&d_work, lwork * sizeof(Number));
    ENFORCE(cudaStatus == hipSuccess);

    // LU factorization of d_A , with partial pivoting and row
    // interchanges ; row i is interchanged with row d_pivot ( i );
    cusolverStatus = libnegf::cusolverDngetrf(
        hcusolver, n, n, d_LU, n, d_work, d_pivot, d_info
    );

    // use the LU factorization to solve the system d_LU * x = d_Ainv ;
    // the solution overwrites d_Ainv
    cusolverStatus = libnegf::cusolverDngetrs(
        hcusolver, HIPBLAS_OP_N, n, n, d_LU, n, d_pivot, d_Ainv, n, d_info
    );
    ENFORCE(cusolverStatus == HIPSOLVER_STATUS_SUCCESS);

    int info_gpu;
    // d_info -> info_gpu
    cudaStatus =
        hipMemcpy(&info_gpu, d_info, sizeof(int), hipMemcpyDeviceToHost);
    ENFORCE(cudaStatus == hipSuccess);

    cudaStatus = hipFree(d_pivot);
    ENFORCE(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_info);
    ENFORCE(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_work);
    ENFORCE(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_LU);
    ENFORCE(cudaStatus == hipSuccess);

    return cudaStatus;
}

extern "C" int cu_Cinverse(
    hipblasHandle_t hcublas, hipsolverHandle_t hcusolver, hipComplex* d_A,
    hipComplex* d_Ainv, size_t n
) {
    return inverse(hcublas, hcusolver, d_A, d_Ainv, n);
}

extern "C" int cu_Zinverse(
    hipblasHandle_t hcublas, hipsolverHandle_t hcusolver, hipDoubleComplex* d_A,
    hipDoubleComplex* d_Ainv, size_t n
) {
    return inverse(hcublas, hcusolver, d_A, d_Ainv, n);
}

template<typename Number>
int cu_kernelsum(
    Number* d_C, Number* alpha, Number* d_A, Number* beta, Number* d_B,
    size_t size
) {
    auto num_blocks = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;

    addKernel<<<num_blocks, BLOCK_SIZE>>>(d_C, *alpha, d_A, *beta, d_B, size);
    ENFORCE(hipGetLastError() == hipSuccess);

    return 0;
}


extern "C" int cu_Ckernelsum(
    hipComplex* d_C, hipComplex* alpha, hipComplex* d_A, hipComplex* beta,
    hipComplex* d_B, size_t size
) {
    return cu_kernelsum(d_C, alpha, d_A, beta, d_B, size);
}

extern "C" int cu_Zkernelsum(
    hipDoubleComplex* d_C, hipDoubleComplex* alpha, hipDoubleComplex* d_A,
    hipDoubleComplex* beta, hipDoubleComplex* d_B, size_t size
) {
    return cu_kernelsum(d_C, alpha, d_A, beta, d_B, size);
}


/**
 * Computes the sum of α op(A) + β op(B), where op(.) indicates if the matrix
 * or its complex-conjugate is used.
 *
 * The possible options are:
 * * `dagger == 0`: compute C ≔ α A + β B
 * * `dagger == 1`: compute C ≔ α A^* + β B
 * * `dagger == 2`: compute C ≔ α A + B^*
 *
 * @param[in] m The number of rows of op(A) and op(B).
 * @param[in] n The number of columns of op(A) and op(B).
 * @param[in] dagger A shorthand for various combinations of op(A), op(B).
 */
template<typename Number>
int cu_matsum(
    hipblasHandle_t hcublas, size_t m, size_t n, const Number* alpha,
    const Number* d_A, const Number* beta, const Number* d_B, Number* d_C,
    int dagger
) {
    ENFORCE(d_A);
    ENFORCE(d_B);
    ENFORCE(d_C);
    ENFORCE(d_A != d_C || dagger == 0 || dagger == 2);
    ENFORCE(d_B != d_C || dagger == 1);

    hipblasStatus_t err;
    if(dagger == 0) {
        err = libnegf::cublasGeam(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, alpha, d_A, m, beta, d_B,
            m, d_C, m
        );
    } else if(dagger == 1) {
        err = libnegf::cublasGeam(
            hcublas, HIPBLAS_OP_C, HIPBLAS_OP_N, m, n, alpha, d_A, n, beta, d_B,
            m, d_C, m
        );
    } else if(dagger == 2) {
        err = libnegf::cublasGeam(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_C, m, n, alpha, d_A, m, beta, d_B,
            n, d_C, m
        );
    } else {
        std::fprintf(stderr, "expected dagger in [0, 1, 2], got %d\n", dagger);
        std::exit(EXIT_FAILURE);
    }

    return err;
}

extern "C" int cu_Cmatsum(
    hipblasHandle_t hcublas, size_t m, size_t n, const hipComplex* alpha,
    const hipComplex* d_A, const hipComplex* beta, const hipComplex* d_B,
    hipComplex* d_C, int dagger
) {
    return cu_matsum(hcublas, m, n, alpha, d_A, beta, d_B, d_C, dagger);
}

extern "C" int cu_Zmatsum(
    hipblasHandle_t hcublas, size_t m, size_t n, const hipDoubleComplex* alpha,
    const hipDoubleComplex* d_A, const hipDoubleComplex* beta,
    const hipDoubleComplex* d_B, hipDoubleComplex* d_C, int dagger
) {
    return cu_matsum(hcublas, m, n, alpha, d_A, beta, d_B, d_C, dagger);
}

extern "C" int cu_Cinitmat(hipComplex* d_A, size_t nrow) {
    ENFORCE(d_A);
    auto size = nrow * nrow;
    auto num_blocks = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;

    initKernel<<<num_blocks, BLOCK_SIZE>>>(d_A, nrow);
    ENFORCE(hipGetLastError() == hipSuccess);

    return 0;
}

extern "C" int cu_Zinitmat(hipDoubleComplex* d_A, size_t nrow) {
    ENFORCE(d_A);
    auto size = nrow * nrow;
    auto num_blocks = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;

    initKernel<<<num_blocks, BLOCK_SIZE>>>(d_A, nrow);
    ENFORCE(hipGetLastError() == hipSuccess);

    return 0;
}


template<typename Number, typename Real = typename get_real<Number>::type>
Real trace(
    hipblasHandle_t hcublas, Number* d_A, size_t nrow, void* h_mask,
    int mask_present
) {
    auto size = nrow * nrow;
    auto num_blocks = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    Real* d_work;
    hipError_t cudaStatus = hipMalloc((void**)&d_work, nrow * sizeof(Real));
    Real* d_iden;
    cudaStatus = hipMalloc((void**)&d_iden, nrow * sizeof(Real));
    ENFORCE(cudaStatus == hipSuccess);
    bool* d_mask;
    cudaStatus = hipMalloc((void**)&d_mask, nrow * sizeof(bool));
    ENFORCE(cudaStatus == hipSuccess);
    if(h_mask) {
        cudaStatus = hipMemcpy(
            d_mask, h_mask, nrow * sizeof(bool), hipMemcpyHostToDevice
        );
        ENFORCE(cudaStatus == hipSuccess);
    }

    initArrayWithOnes<<<num_blocks, BLOCK_SIZE>>>(d_iden, nrow);
    traceKernel<<<num_blocks, BLOCK_SIZE>>>(
        d_A, nrow, d_work, d_mask, mask_present
    );

    Real result;
    hipblasStatus_t err =
        libnegf::cublasDot(hcublas, nrow, d_iden, 1, d_work, 1, &result);
    ENFORCE(err == HIPBLAS_STATUS_SUCCESS);

    cudaStatus = hipFree(d_work);
    cudaStatus = hipFree(d_iden);
    cudaStatus = hipFree(d_mask);

    return result;
}

extern "C" float cu_Ctrace(
    hipblasHandle_t hcublas, hipComplex* d_A, size_t nrow, void* h_mask,
    int mask_present
) {
    return trace(hcublas, d_A, nrow, h_mask, mask_present);
}


extern "C" double cu_Ztrace(
    hipblasHandle_t hcublas, hipDoubleComplex* d_A, size_t nrow, void* h_mask,
    int mask_present
) {
    return trace(hcublas, d_A, nrow, h_mask, mask_present);
}


extern "C" int cu_Cmatcopy(
    hipblasHandle_t hcublas, const hipComplex* d_A, hipComplex* d_B,
    size_t num_elements
) {
    ENFORCE(d_A);
    ENFORCE(d_B);
    ENFORCE(num_elements <= INT_MAX);

    auto err = hipblasCcopy(hcublas, num_elements, d_A, 1, d_B, 1);
    ENFORCE(err == HIPBLAS_STATUS_SUCCESS);
    return err;
}

extern "C" int cu_Zmatcopy(
    hipblasHandle_t hcublas, hipDoubleComplex* d_A, hipDoubleComplex* d_B,
    size_t num_elements
) {
    ENFORCE(d_A);
    ENFORCE(d_B);
    ENFORCE(num_elements <= INT_MAX);

    auto err = hipblasZcopy(hcublas, num_elements, d_A, 1, d_B, 1);
    ENFORCE(err == HIPBLAS_STATUS_SUCCESS);
    return err;
}

extern "C" int
cu_Casum(hipblasHandle_t hcublas, void* d_A, float* summ, size_t n) {
    hipComplex* pdA = (hipComplex*)d_A;

    auto err = hipblasScasum(hcublas, n, pdA, 1, summ);
    ENFORCE(err == HIPBLAS_STATUS_SUCCESS);
    return err;
}

extern "C" int
cu_Zasum(hipblasHandle_t hcublas, void* d_A, double* summ, size_t n) {
    hipDoubleComplex* pdA = (hipDoubleComplex*)d_A;

    auto err = hipblasDzasum(hcublas, n, pdA, 1, summ);
    ENFORCE(err == HIPBLAS_STATUS_SUCCESS);
    return err;
}


template<typename Number>
int decimation(
    hipblasHandle_t hcublas, hipsolverHandle_t hcusolver, Number* h_Go_out,
    Number* h_Ao_in, Number* h_Bo_in, Number* h_Co_in, size_t n, int tf32,
    int* ncyc, float SGFACC
) {
    ENFORCE(h_Go_out);
    ENFORCE(h_Ao_in);
    ENFORCE(h_Bo_in);
    ENFORCE(h_Co_in);
    ENFORCE(tf32 == 0 || tf32 == 1);
    ENFORCE(ncyc);
    ENFORCE(SGFACC > 0.0);

    auto num_elements = n * n;
    auto num_blocks = (num_elements + BLOCK_SIZE - 1) / BLOCK_SIZE;

    using Real = typename get_real<Number>::type;
    auto one = Number{Real{1}};
    auto mone = Number{-Real{1}};
    auto zero = Number{Real{0}};

    Number* d_Ao;
    hipError_t cudaStatus =
        hipMalloc((void**)&d_Ao, num_elements * sizeof(Number));
    ENFORCE(cudaStatus == hipSuccess);

    Number* d_Bo;
    cudaStatus = hipMalloc((void**)&d_Bo, num_elements * sizeof(Number));
    ENFORCE(cudaStatus == hipSuccess);

    Number* d_Co;
    cudaStatus = hipMalloc((void**)&d_Co, num_elements * sizeof(Number));
    ENFORCE(cudaStatus == hipSuccess);

    cudaStatus = hipMemcpy(
        d_Ao, h_Ao_in, n * n * sizeof(Number), hipMemcpyHostToDevice
    );
    ENFORCE(cudaStatus == hipSuccess);
    cudaStatus = hipMemcpy(
        d_Bo, h_Bo_in, n * n * sizeof(Number), hipMemcpyHostToDevice
    );
    ENFORCE(cudaStatus == hipSuccess);
    cudaStatus = hipMemcpy(
        d_Co, h_Co_in, n * n * sizeof(Number), hipMemcpyHostToDevice
    );
    ENFORCE(cudaStatus == hipSuccess);

    hipblasStatus_t hipblasStatus_t =
        hipblasSetPointerMode(hcublas, HIPBLAS_POINTER_MODE_HOST);
    ENFORCE(hipblasStatus_t == hipSuccess);

    if(tf32 == 1) {
        hipblasStatus_t = hipblasSetMathMode(hcublas, HIPBLAS_TENSOR_OP_MATH);
        ENFORCE(hipblasStatus_t == hipSuccess);
    }

    Number* d_Ao_s;
    cudaStatus = hipMalloc((void**)&d_Ao_s, n * n * sizeof(Number));
    ENFORCE(cudaStatus == hipSuccess);
    Number* d_C1;
    cudaStatus = hipMalloc((void**)&d_C1, n * n * sizeof(Number));
    ENFORCE(cudaStatus == hipSuccess);
    Number* d_Go;
    cudaStatus = hipMalloc((void**)&d_Go, n * n * sizeof(Number));
    ENFORCE(cudaStatus == hipSuccess);
    int* d_pivot;
    cudaStatus = hipMalloc((void**)&d_pivot, n * sizeof(int));
    ENFORCE(cudaStatus == hipSuccess);
    Number* d_T;
    cudaStatus = hipMalloc((void**)&d_T, n * n * sizeof(Number));
    ENFORCE(cudaStatus == hipSuccess);
    Number* d_Self;
    cudaStatus = hipMalloc((void**)&d_Self, n * n * sizeof(Number));
    ENFORCE(cudaStatus == hipSuccess);
    int* d_info;
    cudaStatus = hipMalloc((void**)&d_info, sizeof(int));
    ENFORCE(cudaStatus == hipSuccess);

    int lwork;
    hipsolverStatus_t cusolverStatus =
        libnegf::cusolverDngetrf_bufferSize(hcusolver, n, n, d_Self, n, &lwork);
    ENFORCE(cusolverStatus == hipSuccess);
    Number* d_work;
    cudaStatus = hipMalloc((void**)&d_work, lwork * sizeof(Number));
    ENFORCE(cudaStatus == hipSuccess);

    hipblasStatus_t = libnegf::cublasCopy(hcublas, n * n, d_Ao, 1, d_Ao_s, 1);
    ENFORCE(hipblasStatus_t == hipSuccess);

    bool okCo = false;
    for(int i1 = 1; i1 <= 300; i1++) {
        *ncyc = i1;

        initKernel<<<num_blocks, BLOCK_SIZE>>>(d_Go, n);

        hipblasStatus_t = libnegf::cublasCopy(hcublas, n * n, d_Ao, 1, d_Self, 1);
        ENFORCE(hipblasStatus_t == hipSuccess);

        cusolverStatus = libnegf::cusolverDngetrf(
            hcusolver, n, n, d_Self, n, d_work, d_pivot, d_info
        );
        ENFORCE(cusolverStatus == hipSuccess);
        cusolverStatus = libnegf::cusolverDngetrs(
            hcusolver, HIPBLAS_OP_N, n, n, d_Self, n, d_pivot, d_Go, n, d_info
        );
        ENFORCE(cusolverStatus == hipSuccess);

        hipblasStatus_t = libnegf::cublasGemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &one, d_Go, n, d_Co, n,
            &zero, d_T, n
        );
        ENFORCE(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = libnegf::cublasGemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &one, d_Co, n, d_T, n,
            &zero, d_C1, n
        );
        ENFORCE(hipblasStatus_t == hipSuccess);

        Real summ;
        hipblasStatus_t = libnegf::cublasAsum(hcublas, n * n, d_C1, 1, &summ);
        ENFORCE(hipblasStatus_t == hipSuccess);
        // printf("loop it= %d , summ= %f \n ", i1, summ);

        if(summ <= SGFACC) {
            if(okCo) {
                break;
            } else {
                okCo = true;
            }
        } else {
            okCo = false;
        }

        hipblasStatus_t = libnegf::cublasGemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &one, d_Bo, n, d_T, n,
            &zero, d_Self, n
        );
        ENFORCE(hipblasStatus_t == hipSuccess);

        hipblasStatus_t =
            libnegf::cublasAxpy(hcublas, n * n, &mone, d_Self, 1, d_Ao_s, 1);
        ENFORCE(hipblasStatus_t == hipSuccess);
        hipblasStatus_t =
            libnegf::cublasAxpy(hcublas, n * n, &mone, d_Self, 1, d_Ao, 1);
        ENFORCE(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = libnegf::cublasGemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &one, d_Go, n, d_Bo, n,
            &zero, d_T, n
        );
        ENFORCE(hipblasStatus_t == hipSuccess);
        hipblasStatus_t = libnegf::cublasGemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &mone, d_Co, n, d_T, n,
            &one, d_Ao, n
        );
        ENFORCE(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = libnegf::cublasCopy(hcublas, n * n, d_C1, 1, d_Co, 1);
        ENFORCE(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = libnegf::cublasGemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &one, d_Bo, n, d_T, n,
            &zero, d_C1, n
        );
        ENFORCE(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = libnegf::cublasCopy(hcublas, n * n, d_C1, 1, d_Bo, 1);
        ENFORCE(hipblasStatus_t == hipSuccess);
    }

    initKernel<<<num_blocks, BLOCK_SIZE>>>(d_Go, n);
    hipblasStatus_t = libnegf::cublasCopy(hcublas, n * n, d_Ao_s, 1, d_Self, 1);
    ENFORCE(hipblasStatus_t == hipSuccess);
    cusolverStatus = libnegf::cusolverDngetrf(
        hcusolver, n, n, d_Self, n, d_work, d_pivot, d_info
    );
    ENFORCE(cusolverStatus == hipSuccess);
    cusolverStatus = libnegf::cusolverDngetrs(
        hcusolver, HIPBLAS_OP_N, n, n, d_Self, n, d_pivot, d_Go, n, d_info
    );
    ENFORCE(cusolverStatus == hipSuccess);

    cudaStatus = hipMemcpy(
        h_Go_out, d_Go, n * n * sizeof(Number), hipMemcpyDeviceToHost
    );
    ENFORCE(cudaStatus == hipSuccess);

    cudaStatus = hipFree(d_pivot);
    ENFORCE(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_info);
    ENFORCE(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Ao);
    ENFORCE(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Bo);
    ENFORCE(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Co);
    ENFORCE(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Go);
    ENFORCE(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Ao_s);
    ENFORCE(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_C1);
    ENFORCE(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_T);
    ENFORCE(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Self);
    ENFORCE(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_work);
    ENFORCE(cudaStatus == hipSuccess);

    return cudaStatus;
}


extern "C" int cu_Cdecimation(
    hipblasHandle_t hcublas, hipsolverHandle_t hcusolver, hipComplex* h_Go_out,
    hipComplex* h_Ao_in, hipComplex* h_Bo_in, hipComplex* h_Co_in, size_t n,
    int tf32, int* ncyc, float SGFACC
) {
    return decimation(
        hcublas, hcusolver, h_Go_out, h_Ao_in, h_Bo_in, h_Co_in, n, tf32, ncyc,
        SGFACC
    );
}

extern "C" int cu_Zdecimation(
    hipblasHandle_t hcublas, hipsolverHandle_t hcusolver,
    hipDoubleComplex* h_Go_out, hipDoubleComplex* h_Ao_in,
    hipDoubleComplex* h_Bo_in, hipDoubleComplex* h_Co_in, size_t n, int tf32,
    int* ncyc, double SGFACC
) {
    return decimation(
        hcublas, hcusolver, h_Go_out, h_Ao_in, h_Bo_in, h_Co_in, n, tf32, ncyc,
        SGFACC
    );
}


extern "C" int cu_meminfo(size_t* freemem, size_t* totalmem) {
    hipError_t cudaStatus;
    cudaStatus = hipDeviceSynchronize();
    cudaStatus = hipMemGetInfo(freemem, totalmem);
    ENFORCE(cudaStatus == hipSuccess);
    return cudaStatus;
}
