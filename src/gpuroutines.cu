/*!!--------------------------------------------------------------------------!
 *!! libNEGF: a general library for Non-Equilibrium Greens functions.         !
 *!! Copyright (C) 2012 - 2026                                                !
 *!!                                                                          !
 *!! This file is part of libNEGF: a library for                              !
 *!! Non Equilibrium Green's Function calculation                             !
 *!!                                                                          !
 *!! Developers: Alessandro Pecchia, Daniele Soccodato                        !
 *!! Former Contributors: Gabriele Penazzi, Luca Latessa, Aldo Di Carlo       !
 *!!                                                                          !
 *!! libNEGF is free software: you can redistribute it and/or modify          !
 *!! it under the terms of the GNU Lesse General Public License as published  !
 *!! by the Free Software Foundation, either version 3 of the License, or     !
 *!! (at your option) any later version.                                      !
 *!!                                                                          !
 *!!  You should have received a copy of the GNU Lesser General Public        !
 *!!  License along with libNEGF.  If not, see                                !
 *!!  <http://www.gnu.org/licenses/>.                                         !
 *!!--------------------------------------------------------------------------!
 */

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <hipsolver.h>

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 1024
#define TILE_DIM 32
#define BLOCK_ROWS 8

__global__ void CaddKernel(
    hipComplex* c, const hipComplex alpha, const hipComplex* a,
    const hipComplex beta, const hipComplex* b, size_t size
) {
    assert(c);
    assert(a);
    assert(b);

    auto i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < size) {
        c[i].x = (alpha.x * a[i].x - alpha.y * a[i].y) +
                 (beta.x * b[i].x - beta.y * b[i].y);
        c[i].y = (alpha.x * a[i].y + alpha.y * a[i].x) +
                 (beta.x * b[i].y + beta.y * b[i].x);
    }
}

__global__ void ZaddKernel(
    hipDoubleComplex* c, const hipDoubleComplex alpha, const hipDoubleComplex* a,
    const hipDoubleComplex beta, const hipDoubleComplex* b, size_t size
) {
    assert(c);
    assert(a);
    assert(b);

    auto i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < size) {
        c[i].x = (alpha.x * a[i].x - alpha.y * a[i].y) +
                 (beta.x * b[i].x - beta.y * b[i].y);
        c[i].y = (alpha.x * a[i].y + alpha.y * a[i].x) +
                 (beta.x * b[i].y + beta.y * b[i].x);
    }
}

/*
__global__ void hermitian(cuComplex *odata, const cuComplex *idata)
{
  __shared__ cuComplex tile[TILE_DIM][TILE_DIM];

  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
  {
     odata[(y+j)*width + x].x = tile[threadIdx.x][threadIdx.y + j].x;
     odata[(y+j)*width + x].y = -tile[threadIdx.x][threadIdx.y + j].y;
  }
}
*/
__global__ void CinitKernel(hipComplex* a, size_t nrow) {
    assert(a);

    auto size = nrow * nrow;
    auto i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i < size) {
        if(i % (nrow + 1) == 0) {
            a[i].x = 1.0;
            a[i].y = 0.0;
        } else {
            a[i].x = 0.0;
            a[i].y = 0.0;
        }
    }
}

__global__ void ZinitKernel(hipDoubleComplex* a, size_t nrow) {
    assert(a);

    auto size = nrow * nrow;
    auto i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i < size) {
        if(i % (nrow + 1) == 0) {
            a[i].x = 1.0;
            a[i].y = 0.0;
        } else {
            a[i].x = 0.0;
            a[i].y = 0.0;
        }
    }
}

__global__ void DinitKernel(double* a, size_t nrow) {
    assert(a);

    auto i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i < nrow) {
        a[i] = 1.0;
    }
}

__global__ void SinitKernel(float* a, size_t nrow) {
    assert(a);

    auto i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i < nrow) {
        a[i] = 1.0;
    }
}

__global__ void CtraceKernel(
    hipComplex* a, size_t nrow, float* trace, bool* mask, int mask_present
) {
    assert(a);
    assert(trace);
    assert(mask || mask_present == 0);
    assert(mask_present == 0 || mask_present == 1);

    auto size = nrow * nrow;
    auto i = blockDim.x * blockIdx.x + threadIdx.x;

    if(mask_present == 0) {
        if(i < size) {
            if(i % (nrow + 1) == 0) {
                trace[i % nrow] = a[i].x;
            }
        }
    }
    if(mask_present == 1) {
        if(i < size) {
            if(i % (nrow + 1) == 0) {
                if(mask[i % nrow]) {
                    trace[i % nrow] = a[i].x;
                } else {
                    trace[i % nrow] = 0.0;
                }
            }
        }
    }
}

__global__ void ZtraceKernel(
    hipDoubleComplex* a, size_t nrow, double* trace, bool* mask, int mask_present
) {
    assert(a);
    assert(trace);
    assert(mask || mask_present == 0);
    assert(mask_present == 0 || mask_present == 1);

    auto size = nrow * nrow;
    auto i = blockDim.x * blockIdx.x + threadIdx.x;

    if(mask_present == 0) {
        if(i < size) {
            if(i % (nrow + 1) == 0) {
                trace[i % nrow] = a[i].x;
            }
        }
    }
    if(mask_present == 1) {
        if(i < size) {
            if(i % (nrow + 1) == 0) {
                if(mask[i % nrow]) {
                    trace[i % nrow] = a[i].x;
                } else {
                    trace[i % nrow] = 0.0;
                }
            }
        }
    }
}

/*~-~-~-~-~-~-~-~-~-~-~-~-~-~ DATA MOVEMENT  ROUTINES
 * -~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-*/

extern "C" int cu_createMat(void** d_A, size_t bytecount) {
    assert(d_A);
    hipError_t err = hipMalloc(d_A, bytecount);
    //printf("create mat at GPU Address: %p \n",*d_A);
    return err;
}

extern "C" int cu_copyMatH2D(void* h_A, void* d_A, size_t bytecount) {
    assert(h_A);
    assert(d_A);
    // printf("copy %p to %p\n",h_A,d_A);
    hipError_t err = hipMemcpy(d_A, h_A, bytecount, hipMemcpyHostToDevice);
    return err;
}

extern "C" int cu_copyMatD2H(void* h_A, void* d_A, size_t bytecount) {
    assert(h_A);
    assert(d_A);

    hipError_t err = hipMemcpy(h_A, d_A, bytecount, hipMemcpyDeviceToHost);
    return err;
}

extern "C" int cu_deleteMat(void** d_A) {
    int stat = 0;
    if(*d_A != NULL) {
        stat = hipFree(*d_A);
        *d_A = NULL;
    }
    return stat;
}

/*~-~-~-~-~-~-~-~-~-~-~-~-~-~ INIT/FINAL ROUTINES
 * -~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-*/

extern "C" int cu_cudaGetDeviceCount(int* count) {
    assert(count);
    hipError_t err = hipGetDeviceCount(count);
    assert(err == hipSuccess);
    return err;
}

extern "C" int cu_cudaGetDeviceProperties(int device) {
    hipDeviceProp_t prop;
    hipError_t err = hipGetDeviceProperties(&prop, device);
    assert(err == hipSuccess);

    printf(" Found GPU: Device Name: %s\n", prop.name);
    printf(" TotalMemory: %lu\n", (unsigned long)prop.totalGlobalMem);
    printf(" Shared per block: %lu\n", (unsigned long)prop.sharedMemPerBlock);

    return err;
}

extern "C" int cu_cudaSetDevice(int count) {
    hipError_t err = hipSetDevice(count);
    assert(err == hipSuccess);
    return err;
}

extern "C" int cu_cublasInit(hipblasHandle_t* hcublas) {
    assert(hcublas);
    hipblasStatus_t err = hipblasCreate(hcublas);
    assert(err == HIPBLAS_STATUS_SUCCESS);
    if(err != HIPBLAS_STATUS_SUCCESS) {
        printf("cublas create error: %d\n", err);
    }
    // printf("hcublas Addr: %p \n",*hcublas);
    return err;
}

extern "C" int cu_cublasFinalize(hipblasHandle_t hcublas) {
    hipblasStatus_t err = hipblasDestroy(hcublas);
    assert(err == HIPBLAS_STATUS_SUCCESS);
    return err;
}

extern "C" int cu_cusolverInit(hipsolverHandle_t* hcusolver) {
    assert(hcusolver);
    hipsolverStatus_t err = hipsolverDnCreate(hcusolver);
    assert(err == hipSuccess);
    if(err != 0) {
        printf("cusolver create error: %d\n", err);
    }
    // printf("hcusolver Addr: %p \n",*hcusolver);
    return err;
}

extern "C" int cu_cusolverFinalize(hipsolverHandle_t hcusolver) {
    hipsolverStatus_t err = hipsolverDnDestroy(hcusolver);
    assert(err == hipSuccess);
    return err;
}

/*~-~-~-~-~-~-~-~-~-~-~-~-~-~ MATRIX ROUTINES
 * -~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-*/
/* C = alpha op(A) op(B) + beta C
 * m: #rows of op(A)
 * n: #cols of op(B)
 * k: #cols of op(A) = #rows of op(B)
 */
extern "C" int cu_CmultMat(
    hipblasHandle_t hcublas, size_t m, size_t n, size_t k,
    const hipComplex* alpha, const void* d_A, const void* d_B,
    const hipComplex* beta, void* d_C, int dagger
) {
    assert(alpha);
    assert(d_A);
    assert(d_B);
    assert(beta);
    assert(dagger == 0 || dagger == 1 || dagger == 2);

    hipComplex* pdA = (hipComplex*)d_A;
    hipComplex* pdB = (hipComplex*)d_B;
    hipComplex* pdC = (hipComplex*)d_C;

    hipblasStatus_t err;
    if(dagger == 0) {
        err = hipblasCgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, pdA, m, pdB, k,
            beta, pdC, m
        );
    } else if(dagger == 1) {
        err = hipblasCgemm(
            hcublas, HIPBLAS_OP_C, HIPBLAS_OP_N, m, n, k, alpha, pdA, k, pdB, k,
            beta, pdC, m
        );
    } else if(dagger == 2) {
        err = hipblasCgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_C, m, n, k, alpha, pdA, m, pdB, n,
            beta, pdC, m
        );
    }
    assert(err == HIPBLAS_STATUS_SUCCESS);
    return err;
}

//  C = alpha op(A) op(B) + beta C
// op(A):  m x k
// op(B):  k x n
//     C:  m x n
extern "C" int cu_ZmultMat(
    hipblasHandle_t hcublas, size_t m, size_t n, size_t k,
    const hipDoubleComplex* alpha, const void* d_A, const void* d_B,
    const hipDoubleComplex* beta, void* d_C, int dagger
) {
    assert(alpha);
    assert(d_A);
    assert(d_B);
    assert(beta);
    assert(dagger == 0 || dagger == 1 || dagger == 2);

    hipDoubleComplex* pdA = (hipDoubleComplex*)d_A;
    hipDoubleComplex* pdB = (hipDoubleComplex*)d_B;
    hipDoubleComplex* pdC = (hipDoubleComplex*)d_C;
    hipblasStatus_t err;

    if(dagger == 0) {
        err = hipblasZgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, pdA, m, pdB, k,
            beta, pdC, m
        );
    } else if(dagger == 1) {
        err = hipblasZgemm(
            hcublas, HIPBLAS_OP_C, HIPBLAS_OP_N, m, n, k, alpha, pdA, k, pdB, k,
            beta, pdC, m
        );
    } else if(dagger == 2) {
        err = hipblasZgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_C, m, n, k, alpha, pdA, m, pdB, n,
            beta, pdC, m
        );
    }
    assert(err == HIPBLAS_STATUS_SUCCESS);
    return err;
}

extern "C" int cu_Cinverse(
    hipblasHandle_t hcublas, hipsolverHandle_t hcusolver, void* d_A,
    void* d_Ainv, size_t n
) {
    assert(hcusolver);
    assert(d_A);
    assert(d_Ainv);

    // compute buffer size and prep . memory
    hipComplex* pdA = (hipComplex*)d_A;
    int lwork;
    hipsolverStatus_t cusolverStatus =
        hipsolverDnCgetrf_bufferSize(hcusolver, n, n, pdA, n, &lwork);
    assert(cusolverStatus == HIPSOLVER_STATUS_SUCCESS);

    // prepare memory on the device
    hipComplex* d_LU;
    hipError_t cudaStatus =
        hipMalloc((void**)&d_LU, n * n * sizeof(hipComplex));
    assert(cudaStatus == hipSuccess);

    int* d_pivot;
    cudaStatus = hipMalloc((void**)&d_pivot, n * sizeof(int));
    assert(cudaStatus == hipSuccess);
    int* d_info;
    cudaStatus = hipMalloc((void**)&d_info, sizeof(int));
    assert(cudaStatus == hipSuccess);
    // copy d_LU <- pdA
    hipblasStatus_t hipblasStatus_t = hipblasCcopy(hcublas, n * n, pdA, 1, d_LU, 1);
    assert(hipblasStatus_t == HIPBLAS_STATUS_SUCCESS);

    hipComplex* d_work;
    cudaStatus = hipMalloc((void**)&d_work, lwork * sizeof(hipComplex));
    assert(cudaStatus == hipSuccess);

    // LU factorization of d_A , with partial pivoting and row
    // interchanges ; row i is interchanged with row d_pivot ( i );
    cusolverStatus =
        hipsolverDnCgetrf(hcusolver, n, n, d_LU, n, d_work, d_pivot, d_info);

    // use the LU factorization to solve the system d_LU * x = d_Ainv ;
    // the solution overwrites d_Ainv
    hipComplex* pdAinv = (hipComplex*)d_Ainv;
    cusolverStatus = hipsolverDnCgetrs(
        hcusolver, HIPBLAS_OP_N, n, n, d_LU, n, d_pivot, pdAinv, n, d_info
    );
    assert(cusolverStatus == HIPSOLVER_STATUS_SUCCESS);

    int info_gpu;
    // d_info -> info_gpu
    cudaStatus =
        hipMemcpy(&info_gpu, d_info, sizeof(int), hipMemcpyDeviceToHost);
    assert(cudaStatus == hipSuccess);

    cudaStatus = hipFree(d_pivot);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_info);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_work);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_LU);
    assert(cudaStatus == hipSuccess);

    return cudaStatus;
}

extern "C" int cu_Zinverse(
    hipblasHandle_t hcublas, hipsolverHandle_t hcusolver, void* d_A,
    void* d_Ainv, size_t n
) {
    assert(hcusolver);
    assert(d_A);
    assert(d_Ainv);

    // compute buffer size and prep . memory
    hipDoubleComplex* pdA = (hipDoubleComplex*)d_A;
    int lwork;
    hipsolverStatus_t cusolverStatus =
        hipsolverDnZgetrf_bufferSize(hcusolver, n, n, pdA, n, &lwork);
    assert(cusolverStatus == HIPSOLVER_STATUS_SUCCESS);

    // prepare memory on the device
    hipDoubleComplex* d_LU;
    hipError_t cudaStatus =
        hipMalloc((void**)&d_LU, n * n * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);

    int* d_pivot;
    cudaStatus = hipMalloc((void**)&d_pivot, n * sizeof(int));
    assert(cudaStatus == hipSuccess);
    int* d_info;
    cudaStatus = hipMalloc((void**)&d_info, sizeof(int));
    assert(cudaStatus == hipSuccess);
    // copy d_LU <- pdA
    hipblasStatus_t hipblasStatus_t = hipblasZcopy(hcublas, n * n, pdA, 1, d_LU, 1);
    assert(hipblasStatus_t == HIPBLAS_STATUS_SUCCESS);

    hipDoubleComplex* d_work;
    cudaStatus = hipMalloc((void**)&d_work, lwork * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);

    // LU factorization of d_A , with partial pivoting and row
    // interchanges ; row i is interchanged with row d_pivot ( i );
    cusolverStatus =
        hipsolverDnZgetrf(hcusolver, n, n, d_LU, n, d_work, d_pivot, d_info);

    // use the LU factorization to solve the system d_LU * x = d_Ainv ;
    // the solution overwrites d_Ainv
    hipDoubleComplex* pdAinv = (hipDoubleComplex*)d_Ainv;
    cusolverStatus = hipsolverDnZgetrs(
        hcusolver, HIPBLAS_OP_N, n, n, d_LU, n, d_pivot, pdAinv, n, d_info
    );
    assert(cusolverStatus == HIPSOLVER_STATUS_SUCCESS);

    int info_gpu;
    // d_info -> info_gpu
    cudaStatus =
        hipMemcpy(&info_gpu, d_info, sizeof(int), hipMemcpyDeviceToHost);
    assert(cudaStatus == hipSuccess);

    cudaStatus = hipFree(d_pivot);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_info);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_work);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_LU);
    assert(cudaStatus == hipSuccess);

    return cudaStatus;
}

extern "C" int cu_Cmatsum(
    hipblasHandle_t hcublas, size_t m, size_t n, hipComplex* alpha, void* d_A,
    hipComplex* beta, void* d_B, void* d_C, int dagger
) {
    // m number of rows of matrix op(A) and C
    // n number of columns of matrix op(B) and C
    hipComplex* pdA = (hipComplex*)d_A;
    hipComplex* pdB = (hipComplex*)d_B;
    hipComplex* pdC = (hipComplex*)d_C;

    hipblasStatus_t err;
    if(dagger == 0) {
        err = hipblasCgeam(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, alpha, pdA, m, beta, pdB,
            m, pdC, m
        );
    }
    if(dagger == 1) {
        err = hipblasCgeam(
            hcublas, HIPBLAS_OP_C, HIPBLAS_OP_N, m, n, alpha, pdA, n, beta, pdB,
            m, pdC, m
        );
    }
    if(dagger == 2) {
        err = hipblasCgeam(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_C, m, n, alpha, pdA, m, beta, pdB,
            n, pdC, m
        );
    }
    return err;
}

extern "C" int cu_Zmatsum(
    hipblasHandle_t hcublas, size_t m, size_t n, hipDoubleComplex* alpha,
    void* d_A, hipDoubleComplex* beta, void* d_B, void* d_C, int dagger
) {
    // m number of rows of matrix op(A) and C
    // n number of columns of matrix op(B) and C
    hipDoubleComplex* pdA = (hipDoubleComplex*)d_A;
    hipDoubleComplex* pdB = (hipDoubleComplex*)d_B;
    hipDoubleComplex* pdC = (hipDoubleComplex*)d_C;

    hipblasStatus_t err;
    if(dagger == 0) {
        err = hipblasZgeam(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, alpha, pdA, m, beta, pdB,
            m, pdC, m
        );
    }
    if(dagger == 1) {
        err = hipblasZgeam(
            hcublas, HIPBLAS_OP_C, HIPBLAS_OP_N, m, n, alpha, pdA, n, beta, pdB,
            m, pdC, m
        );
    }
    if(dagger == 2) {
        err = hipblasZgeam(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_C, m, n, alpha, pdA, m, beta, pdB,
            n, pdC, m
        );
    }
    return err;
}

extern "C" int cu_Cinitmat(void* d_A, size_t nrow) {
    auto size = nrow * nrow;
    auto num_blocks = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    hipComplex* pdA = (hipComplex*)d_A;

    CinitKernel<<<num_blocks, BLOCK_SIZE>>>(pdA, nrow);
    assert(hipGetLastError() == hipSuccess);

    return 0;
}

extern "C" int cu_Zinitmat(void* d_A, size_t nrow) {
    assert(d_A);
    //printf("d_A: %p %d \n",d_A, d_A);

    auto size = nrow * nrow;
    auto num_blocks = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    hipDoubleComplex* pdA = (hipDoubleComplex*)d_A;

    ZinitKernel<<<num_blocks, BLOCK_SIZE>>>(pdA, nrow);
    assert(hipGetLastError() == hipSuccess);

    return 0;
}

extern "C" float cu_Ctrace(
    hipblasHandle_t hcublas, void* d_A, size_t nrow, void* h_mask,
    int mask_present
) {
    hipComplex* pdA = (hipComplex*)d_A;
    auto size = nrow * nrow;
    auto num_blocks = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    float* d_work;
    hipError_t cudaStatus = hipMalloc((void**)&d_work, nrow * sizeof(float));
    float* d_iden;
    cudaStatus = hipMalloc((void**)&d_iden, nrow * sizeof(float));
    assert(cudaStatus == hipSuccess);
    bool* d_mask;
    cudaStatus = hipMalloc((void**)&d_mask, nrow * sizeof(bool));
    assert(cudaStatus == hipSuccess);
    if(h_mask) {
        cudaStatus = hipMemcpy(
            d_mask, h_mask, nrow * sizeof(bool), hipMemcpyHostToDevice
        );
        assert(cudaStatus == hipSuccess);
    }

    SinitKernel<<<num_blocks, BLOCK_SIZE>>>(d_iden, nrow);
    CtraceKernel<<<num_blocks, BLOCK_SIZE>>>(
        pdA, nrow, d_work, d_mask, mask_present
    );

    float result;
    hipblasStatus_t err =
        hipblasSdot(hcublas, nrow, d_iden, 1, d_work, 1, &result);
    assert(err == HIPBLAS_STATUS_SUCCESS);

    cudaStatus = hipFree(d_work);
    cudaStatus = hipFree(d_iden);
    cudaStatus = hipFree(d_mask);

    return result;
}

extern "C" double cu_Ztrace(
    hipblasHandle_t hcublas, void* d_A, size_t nrow, void* h_mask,
    int mask_present
) {
    assert(d_A);
    assert(h_mask || mask_present == 0);
    assert(mask_present == 0 || mask_present == 1);

    hipDoubleComplex* pdA = (hipDoubleComplex*)d_A;
    auto size = nrow * nrow;
    auto num_blocks = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    double* d_work;
    hipError_t cudaStatus = hipMalloc((void**)&d_work, nrow * sizeof(double));
    assert(cudaStatus == hipSuccess);
    double* d_iden;
    cudaStatus = hipMalloc((void**)&d_iden, nrow * sizeof(double));
    assert(cudaStatus == hipSuccess);
    bool* d_mask;
    cudaStatus = hipMalloc((void**)&d_mask, nrow * sizeof(bool));
    assert(cudaStatus == hipSuccess);
    if(h_mask) {
        cudaStatus = hipMemcpy(
            d_mask, h_mask, nrow * sizeof(bool), hipMemcpyHostToDevice
        );
        assert(cudaStatus == hipSuccess);
    }

    DinitKernel<<<num_blocks, BLOCK_SIZE>>>(d_iden, nrow);
    assert(hipGetLastError() == hipSuccess);
    ZtraceKernel<<<num_blocks, BLOCK_SIZE>>>(
        pdA, nrow, d_work, d_mask, mask_present
    );
    assert(hipGetLastError() == hipSuccess);

    double result;
    hipblasStatus_t err =
        hipblasDdot(hcublas, nrow, d_iden, 1, d_work, 1, &result);
    assert(err == HIPBLAS_STATUS_SUCCESS);

    cudaStatus = hipFree(d_work);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_iden);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_mask);
    assert(cudaStatus == hipSuccess);

    return result;
}

extern "C" int
cu_Cmatcopy(hipblasHandle_t hcublas, void* d_A, void* d_B, size_t n) {
    assert(d_A);
    assert(d_B);

    hipComplex* pdA = (hipComplex*)d_A;
    hipComplex* pdB = (hipComplex*)d_B;

    auto err = hipblasCcopy(hcublas, n * n, pdA, 1, pdB, 1);
    assert(err == HIPBLAS_STATUS_SUCCESS);
    return err;
}

extern "C" int
cu_Zmatcopy(hipblasHandle_t hcublas, void* d_A, void* d_B, size_t n) {
    assert(d_A);
    assert(d_B);

    hipDoubleComplex* pdA = (hipDoubleComplex*)d_A;
    hipDoubleComplex* pdB = (hipDoubleComplex*)d_B;

    auto err = hipblasZcopy(hcublas, n, pdA, 1, pdB, 1);
    assert(err == HIPBLAS_STATUS_SUCCESS);
    return err;
}

extern "C" int
cu_Casum(hipblasHandle_t hcublas, void* d_A, float* summ, size_t n) {
    hipComplex* pdA = (hipComplex*)d_A;

    auto err = hipblasScasum(hcublas, n, pdA, 1, summ);
    assert(err == HIPBLAS_STATUS_SUCCESS);
    return err;
}

extern "C" int
cu_Zasum(hipblasHandle_t hcublas, void* d_A, double* summ, size_t n) {
    hipDoubleComplex* pdA = (hipDoubleComplex*)d_A;

    auto err = hipblasDzasum(hcublas, n, pdA, 1, summ);
    assert(err == HIPBLAS_STATUS_SUCCESS);
    return err;
}

extern "C" int cu_Cdecimation(
    hipblasHandle_t hcublas, hipsolverHandle_t hcusolver, void* h_Go_out,
    void* h_Ao_in, void* h_Bo_in, void* h_Co_in, size_t n, int tf32, int* ncyc,
    hipComplex* one, hipComplex* mone, hipComplex* zero, float SGFACC
) {
    assert(h_Go_out);
    assert(h_Ao_in);
    assert(h_Bo_in);
    assert(h_Co_in);
    assert(tf32 == 0 || tf32 == 1);
    assert(ncyc);
    assert(one);
    assert(mone);
    assert(zero);
    assert(SGFACC > 0.0);

    auto num_elements = n * n;
    auto num_blocks = (num_elements + BLOCK_SIZE - 1) / BLOCK_SIZE;

    hipComplex* d_Ao;
    hipError_t cudaStatus =
        hipMalloc((void**)&d_Ao, num_elements * sizeof(hipComplex));
    assert(cudaStatus == hipSuccess);

    hipComplex* d_Bo;
    cudaStatus = hipMalloc((void**)&d_Bo, num_elements * sizeof(hipComplex));
    assert(cudaStatus == hipSuccess);

    hipComplex* d_Co;
    cudaStatus = hipMalloc((void**)&d_Co, num_elements * sizeof(hipComplex));
    assert(cudaStatus == hipSuccess);

    cudaStatus = hipMemcpy(
        d_Ao, h_Ao_in, n * n * sizeof(hipComplex), hipMemcpyHostToDevice
    );
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipMemcpy(
        d_Bo, h_Bo_in, n * n * sizeof(hipComplex), hipMemcpyHostToDevice
    );
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipMemcpy(
        d_Co, h_Co_in, n * n * sizeof(hipComplex), hipMemcpyHostToDevice
    );
    assert(cudaStatus == hipSuccess);

    hipblasStatus_t hipblasStatus_t =
        hipblasSetPointerMode(hcublas, HIPBLAS_POINTER_MODE_HOST);
    assert(hipblasStatus_t == hipSuccess);

    if(tf32 == 1) {
        hipblasStatus_t = hipblasSetMathMode(hcublas, HIPBLAS_TENSOR_OP_MATH);
        assert(hipblasStatus_t == hipSuccess);
    }

    hipComplex* d_Ao_s;
    cudaStatus = hipMalloc((void**)&d_Ao_s, n * n * sizeof(hipComplex));
    assert(cudaStatus == hipSuccess);
    hipComplex* d_C1;
    cudaStatus = hipMalloc((void**)&d_C1, n * n * sizeof(hipComplex));
    assert(cudaStatus == hipSuccess);
    hipComplex* d_Go;
    cudaStatus = hipMalloc((void**)&d_Go, n * n * sizeof(hipComplex));
    assert(cudaStatus == hipSuccess);
    int* d_pivot;
    cudaStatus = hipMalloc((void**)&d_pivot, n * sizeof(int));
    assert(cudaStatus == hipSuccess);
    hipComplex* d_T;
    cudaStatus = hipMalloc((void**)&d_T, n * n * sizeof(hipComplex));
    assert(cudaStatus == hipSuccess);
    hipComplex* d_Self;
    cudaStatus = hipMalloc((void**)&d_Self, n * n * sizeof(hipComplex));
    assert(cudaStatus == hipSuccess);
    int* d_info;
    cudaStatus = hipMalloc((void**)&d_info, sizeof(int));
    assert(cudaStatus == hipSuccess);

    int lwork;
    hipsolverStatus_t cusolverStatus =
        hipsolverDnCgetrf_bufferSize(hcusolver, n, n, d_Self, n, &lwork);
    assert(cusolverStatus == hipSuccess);
    hipComplex* d_work;
    cudaStatus = hipMalloc((void**)&d_work, lwork * sizeof(hipComplex));
    assert(cudaStatus == hipSuccess);

    hipblasStatus_t = hipblasCcopy(hcublas, n * n, d_Ao, 1, d_Ao_s, 1);
    assert(hipblasStatus_t == hipSuccess);

    bool okCo = false;
    for(int i1 = 1; i1 <= 300; i1++) {
        *ncyc = i1;

        CinitKernel<<<num_blocks, BLOCK_SIZE>>>(d_Go, n);

        hipblasStatus_t = hipblasCcopy(hcublas, n * n, d_Ao, 1, d_Self, 1);
        assert(hipblasStatus_t == hipSuccess);

        cusolverStatus = hipsolverDnCgetrf(
            hcusolver, n, n, d_Self, n, d_work, d_pivot, d_info
        );
        assert(cusolverStatus == hipSuccess);
        cusolverStatus = hipsolverDnCgetrs(
            hcusolver, HIPBLAS_OP_N, n, n, d_Self, n, d_pivot, d_Go, n, d_info
        );
        assert(cusolverStatus == hipSuccess);

        hipblasStatus_t = hipblasCgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Go, n, d_Co, n,
            zero, d_T, n
        );
        assert(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = hipblasCgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Co, n, d_T, n,
            zero, d_C1, n
        );
        assert(hipblasStatus_t == hipSuccess);

        float summ;
        hipblasStatus_t = hipblasScasum(hcublas, n * n, d_C1, 1, &summ);
        assert(hipblasStatus_t == hipSuccess);
        // printf("loop it= %d , summ= %f \n ", i1, summ);

        if(summ <= SGFACC) {
            if(okCo) {
                break;
            } else {
                okCo = true;
            }
        } else {
            okCo = false;
        }

        hipblasStatus_t = hipblasCgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Bo, n, d_T, n,
            zero, d_Self, n
        );
        assert(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = hipblasCaxpy(hcublas, n * n, mone, d_Self, 1, d_Ao_s, 1);
        assert(hipblasStatus_t == hipSuccess);
        hipblasStatus_t = hipblasCaxpy(hcublas, n * n, mone, d_Self, 1, d_Ao, 1);
        assert(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = hipblasCgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Go, n, d_Bo, n,
            zero, d_T, n
        );
        assert(hipblasStatus_t == hipSuccess);
        hipblasStatus_t = hipblasCgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, mone, d_Co, n, d_T, n,
            one, d_Ao, n
        );
        assert(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = hipblasCcopy(hcublas, n * n, d_C1, 1, d_Co, 1);
        assert(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = hipblasCgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Bo, n, d_T, n,
            zero, d_C1, n
        );
        assert(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = hipblasCcopy(hcublas, n * n, d_C1, 1, d_Bo, 1);
        assert(hipblasStatus_t == hipSuccess);
    }

    CinitKernel<<<num_blocks, BLOCK_SIZE>>>(d_Go, n);
    hipblasStatus_t = hipblasCcopy(hcublas, n * n, d_Ao_s, 1, d_Self, 1);
    assert(hipblasStatus_t == hipSuccess);
    cusolverStatus =
        hipsolverDnCgetrf(hcusolver, n, n, d_Self, n, d_work, d_pivot, d_info);
    assert(cusolverStatus == hipSuccess);
    cusolverStatus = hipsolverDnCgetrs(
        hcusolver, HIPBLAS_OP_N, n, n, d_Self, n, d_pivot, d_Go, n, d_info
    );
    assert(cusolverStatus == hipSuccess);

    cudaStatus = hipMemcpy(
        h_Go_out, d_Go, n * n * sizeof(hipComplex), hipMemcpyDeviceToHost
    );
    assert(cudaStatus == hipSuccess);

    cudaStatus = hipFree(d_pivot);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_info);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Ao);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Bo);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Co);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Go);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Ao_s);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_C1);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_T);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Self);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_work);
    assert(cudaStatus == hipSuccess);

    return cudaStatus;
}

extern "C" int cu_Zdecimation(
    hipblasHandle_t hcublas, hipsolverHandle_t hcusolver, void* h_Go_out,
    void* h_Ao_in, void* h_Bo_in, void* h_Co_in, size_t n, int tf32, int* ncyc,
    hipDoubleComplex* one, hipDoubleComplex* mone, hipDoubleComplex* zero,
    double SGFACC
) {
    assert(h_Go_out);
    assert(h_Ao_in);
    assert(h_Bo_in);
    assert(h_Co_in);
    assert(tf32 == 0 || tf32 == 1);
    assert(ncyc);
    assert(one);
    assert(mone);
    assert(zero);
    assert(SGFACC > 0.0);

    auto num_elements = n * n;
    auto num_blocks = (num_elements + BLOCK_SIZE - 1) / BLOCK_SIZE;

    hipDoubleComplex* d_Ao;
    hipError_t cudaStatus =
        hipMalloc((void**)&d_Ao, num_elements * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);

    hipDoubleComplex* d_Bo;
    cudaStatus =
        hipMalloc((void**)&d_Bo, num_elements * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);

    hipDoubleComplex* d_Co;
    cudaStatus =
        hipMalloc((void**)&d_Co, num_elements * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);

    cudaStatus = hipMemcpy(
        d_Ao, h_Ao_in, n * n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice
    );
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipMemcpy(
        d_Bo, h_Bo_in, n * n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice
    );
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipMemcpy(
        d_Co, h_Co_in, n * n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice
    );
    assert(cudaStatus == hipSuccess);

    hipblasStatus_t hipblasStatus_t =
        hipblasSetPointerMode(hcublas, HIPBLAS_POINTER_MODE_HOST);
    assert(hipblasStatus_t == hipSuccess);

    if(tf32 == 1) {
        hipblasStatus_t = hipblasSetMathMode(hcublas, HIPBLAS_TENSOR_OP_MATH);
        assert(hipblasStatus_t == hipSuccess);
    }

    hipDoubleComplex* d_Ao_s;
    cudaStatus = hipMalloc((void**)&d_Ao_s, n * n * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);
    hipDoubleComplex* d_C1;
    cudaStatus = hipMalloc((void**)&d_C1, n * n * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);
    hipDoubleComplex* d_Go;
    cudaStatus = hipMalloc((void**)&d_Go, n * n * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);
    int* d_pivot;
    cudaStatus = hipMalloc((void**)&d_pivot, n * sizeof(int));
    assert(cudaStatus == hipSuccess);
    hipDoubleComplex* d_T;
    cudaStatus = hipMalloc((void**)&d_T, n * n * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);
    hipDoubleComplex* d_Self;
    cudaStatus = hipMalloc((void**)&d_Self, n * n * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);
    int* d_info;
    cudaStatus = hipMalloc((void**)&d_info, sizeof(int));
    assert(cudaStatus == hipSuccess);

    int lwork;
    hipsolverStatus_t cusolverStatus =
        hipsolverDnZgetrf_bufferSize(hcusolver, n, n, d_Self, n, &lwork);
    assert(cusolverStatus == hipSuccess);
    hipDoubleComplex* d_work;
    cudaStatus = hipMalloc((void**)&d_work, lwork * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);

    hipblasStatus_t = hipblasZcopy(hcublas, n * n, d_Ao, 1, d_Ao_s, 1);
    assert(hipblasStatus_t == hipSuccess);

    bool okCo = false;
    for(int i1 = 1; i1 <= 300; i1++) {
        *ncyc = i1;

        ZinitKernel<<<num_blocks, BLOCK_SIZE>>>(d_Go, n);

        hipblasStatus_t = hipblasZcopy(hcublas, n * n, d_Ao, 1, d_Self, 1);
        assert(hipblasStatus_t == hipSuccess);

        cusolverStatus = hipsolverDnZgetrf(
            hcusolver, n, n, d_Self, n, d_work, d_pivot, d_info
        );
        assert(cusolverStatus == hipSuccess);
        cusolverStatus = hipsolverDnZgetrs(
            hcusolver, HIPBLAS_OP_N, n, n, d_Self, n, d_pivot, d_Go, n, d_info
        );
        assert(cusolverStatus == hipSuccess);

        hipblasStatus_t = hipblasZgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Go, n, d_Co, n,
            zero, d_T, n
        );
        assert(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = hipblasZgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Co, n, d_T, n,
            zero, d_C1, n
        );
        assert(hipblasStatus_t == hipSuccess);

        double summ;
        hipblasStatus_t = hipblasDzasum(hcublas, n * n, d_C1, 1, &summ);
        assert(hipblasStatus_t == hipSuccess);
        //printf("loop it= %d , summ= %f \n ", i1, summ);

        if(summ <= SGFACC) {
            if(okCo) {
                break;
            } else {
                okCo = true;
            }
        } else {
            okCo = false;
        }

        hipblasStatus_t = hipblasZgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Bo, n, d_T, n,
            zero, d_Self, n
        );
        assert(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = hipblasZaxpy(hcublas, n * n, mone, d_Self, 1, d_Ao_s, 1);
        assert(hipblasStatus_t == hipSuccess);
        hipblasStatus_t = hipblasZaxpy(hcublas, n * n, mone, d_Self, 1, d_Ao, 1);
        assert(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = hipblasZgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Go, n, d_Bo, n,
            zero, d_T, n
        );
        assert(hipblasStatus_t == hipSuccess);
        hipblasStatus_t = hipblasZgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, mone, d_Co, n, d_T, n,
            one, d_Ao, n
        );
        assert(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = hipblasZcopy(hcublas, n * n, d_C1, 1, d_Co, 1);
        assert(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = hipblasZgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Bo, n, d_T, n,
            zero, d_C1, n
        );
        assert(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = hipblasZcopy(hcublas, n * n, d_C1, 1, d_Bo, 1);
        assert(hipblasStatus_t == hipSuccess);
    }

    ZinitKernel<<<num_blocks, BLOCK_SIZE>>>(d_Go, n);
    hipblasStatus_t = hipblasZcopy(hcublas, n * n, d_Ao_s, 1, d_Self, 1);
    assert(hipblasStatus_t == hipSuccess);
    cusolverStatus =
        hipsolverDnZgetrf(hcusolver, n, n, d_Self, n, d_work, d_pivot, d_info);
    assert(cusolverStatus == hipSuccess);
    cusolverStatus = hipsolverDnZgetrs(
        hcusolver, HIPBLAS_OP_N, n, n, d_Self, n, d_pivot, d_Go, n, d_info
    );
    assert(cusolverStatus == hipSuccess);

    cudaStatus = hipMemcpy(
        h_Go_out, d_Go, n * n * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost
    );
    assert(cudaStatus == hipSuccess);

    cudaStatus = hipFree(d_pivot);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_info);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Ao);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Bo);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Co);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Go);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Ao_s);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_C1);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_T);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Self);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_work);
    assert(cudaStatus == hipSuccess);

    return cudaStatus;
}

extern "C" int cu_meminfo(size_t* freemem, size_t* totalmem) {
    hipError_t cudaStatus;
    cudaStatus = hipDeviceSynchronize();
    cudaStatus = hipMemGetInfo(freemem, totalmem);
    assert(cudaStatus == hipSuccess);
    return cudaStatus;
}
