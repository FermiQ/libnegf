#include "hip/hip_runtime.h"
/*!!--------------------------------------------------------------------------!
 *!! libNEGF: a general library for Non-Equilibrium Greens functions.         !
 *!! Copyright (C) 2012 - 2026                                                !
 *!!                                                                          !
 *!! This file is part of libNEGF: a library for                              !
 *!! Non Equilibrium Green's Function calculation                             !
 *!!                                                                          !
 *!! Developers: Alessandro Pecchia, Daniele Soccodato                        !
 *!! Former Contributors: Gabriele Penazzi, Luca Latessa, Aldo Di Carlo       !
 *!!                                                                          !
 *!! libNEGF is free software: you can redistribute it and/or modify          !
 *!! it under the terms of the GNU Lesse General Public License as published  !
 *!! by the Free Software Foundation, either version 3 of the License, or     !
 *!! (at your option) any later version.                                      !
 *!!                                                                          !
 *!!  You should have received a copy of the GNU Lesser General Public        !
 *!!  License along with libNEGF.  If not, see                                !
 *!!  <http://www.gnu.org/licenses/>.                                         !
 *!!--------------------------------------------------------------------------!
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include <cassert>
#include <climits>
#include <cstddef>
#include <cstdint>
#include <cstdio>
#include <cstdlib>

#include <libnegf/hipblas.h>


constexpr auto BLOCK_SIZE = std::size_t{1024};


template<typename>
struct get_real {};

template<>
struct get_real<hipComplex> {
    using type = float;
};

template<>
struct get_real<hipDoubleComplex> {
    using type = double;
};


template<typename Number>
__global__ void addKernel(
    Number* c, Number alpha, const Number* a, Number beta, const Number* b,
    size_t size
) {
    assert(c);
    assert(a);
    assert(b);

    auto i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < size) {
        c[i].x = (alpha.x * a[i].x - alpha.y * a[i].y) +
                 (beta.x * b[i].x - beta.y * b[i].y);
        c[i].y = (alpha.x * a[i].y + alpha.y * a[i].x) +
                 (beta.x * b[i].y + beta.y * b[i].x);
    }
}


/*
__global__ void hermitian(hipComplex *odata, const hipComplex *idata)
{
  __shared__ hipComplex tile[TILE_DIM][TILE_DIM];

  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
  {
     odata[(y+j)*width + x].x = tile[threadIdx.x][threadIdx.y + j].x;
     odata[(y+j)*width + x].y = -tile[threadIdx.x][threadIdx.y + j].y;
  }
}
*/


template<typename Number>
__global__ void initKernel(Number* a, size_t nrow) {
    assert(a);

    auto size = nrow * nrow;
    auto i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i < size) {
        if(i % (nrow + 1) == 0) {
            a[i].x = 1.0;
            a[i].y = 0.0;
        } else {
            a[i].x = 0.0;
            a[i].y = 0.0;
        }
    }
}


__global__ void DinitKernel(double* a, size_t nrow) {
    assert(a);

    auto i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i < nrow) {
        a[i] = 1.0;
    }
}

__global__ void SinitKernel(float* a, size_t nrow) {
    assert(a);

    auto i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i < nrow) {
        a[i] = 1.0;
    }
}


template<typename Number, typename Real = typename get_real<Number>::type>
__device__ void
traceKernel(Number* a, size_t nrow, Real* trace, bool* mask, int mask_present) {
    assert(a);
    assert(trace);
    assert(mask || mask_present == 0);
    assert(mask_present == 0 || mask_present == 1);

    auto size = nrow * nrow;
    auto i = blockDim.x * blockIdx.x + threadIdx.x;

    if(mask_present == 0) {
        if(i < size) {
            if(i % (nrow + 1) == 0) {
                trace[i % nrow] = a[i].x;
            }
        }
    }
    if(mask_present == 1) {
        if(i < size) {
            if(i % (nrow + 1) == 0) {
                if(mask[i % nrow]) {
                    trace[i % nrow] = a[i].x;
                } else {
                    trace[i % nrow] = 0.0;
                }
            }
        }
    }
}

__global__ void CtraceKernel(
    hipComplex* a, size_t nrow, float* trace, bool* mask, int mask_present
) {
    traceKernel(a, nrow, trace, mask, mask_present);
}

__global__ void ZtraceKernel(
    hipDoubleComplex* a, size_t nrow, double* trace, bool* mask, int mask_present
) {
    traceKernel(a, nrow, trace, mask, mask_present);
}


/*~-~-~-~-~-~-~-~-~-~-~-~-~-~ DATA MOVEMENT  ROUTINES
 * -~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-*/

extern "C" int cu_createMat(void** d_A, size_t bytecount) {
    assert(d_A);
    hipError_t err = hipMalloc(d_A, bytecount);
    //printf("create mat at GPU Address: %p \n",*d_A);
    return err;
}

extern "C" int cu_copyMatH2D(void* h_A, void* d_A, size_t bytecount) {
    assert(h_A);
    assert(d_A);
    // printf("copy %p to %p\n",h_A,d_A);
    hipError_t err = hipMemcpy(d_A, h_A, bytecount, hipMemcpyHostToDevice);
    return err;
}

extern "C" int cu_copyMatD2H(void* h_A, void* d_A, size_t bytecount) {
    assert(h_A);
    assert(d_A);

    hipError_t err = hipMemcpy(h_A, d_A, bytecount, hipMemcpyDeviceToHost);
    return err;
}

extern "C" int cu_deleteMat(void** d_A) {
    int stat = 0;
    if(*d_A != NULL) {
        stat = hipFree(*d_A);
        *d_A = NULL;
    }
    return stat;
}

/*~-~-~-~-~-~-~-~-~-~-~-~-~-~ INIT/FINAL ROUTINES
 * -~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-*/

extern "C" int cu_cudaGetDeviceCount(int* count) {
    assert(count);
    hipError_t err = hipGetDeviceCount(count);
    assert(err == hipSuccess);
    return err;
}

extern "C" int cu_cudaGetDeviceProperties(int device) {
    hipDeviceProp_t prop;
    hipError_t err = hipGetDeviceProperties(&prop, device);
    assert(err == hipSuccess);

    printf(" Found GPU: Device Name: %s\n", prop.name);
    printf(" TotalMemory: %lu\n", (unsigned long)prop.totalGlobalMem);
    printf(" Shared per block: %lu\n", (unsigned long)prop.sharedMemPerBlock);

    return err;
}

extern "C" int cu_cudaSetDevice(int count) {
    hipError_t err = hipSetDevice(count);
    assert(err == hipSuccess);
    return err;
}

extern "C" int cu_cublasInit(hipblasHandle_t* hcublas) {
    assert(hcublas);
    hipblasStatus_t err = hipblasCreate(hcublas);
    assert(err == HIPBLAS_STATUS_SUCCESS);
    if(err != HIPBLAS_STATUS_SUCCESS) {
        printf("cublas create error: %d\n", err);
    }
    // printf("hcublas Addr: %p \n",*hcublas);
    return err;
}

extern "C" int cu_cublasFinalize(hipblasHandle_t hcublas) {
    hipblasStatus_t err = hipblasDestroy(hcublas);
    assert(err == HIPBLAS_STATUS_SUCCESS);
    return err;
}

extern "C" int cu_cusolverInit(hipsolverHandle_t* hcusolver) {
    assert(hcusolver);
    hipsolverStatus_t err = hipsolverDnCreate(hcusolver);
    assert(err == hipSuccess);
    if(err != 0) {
        printf("cusolver create error: %d\n", err);
    }
    // printf("hcusolver Addr: %p \n",*hcusolver);
    return err;
}

extern "C" int cu_cusolverFinalize(hipsolverHandle_t hcusolver) {
    hipsolverStatus_t err = hipsolverDnDestroy(hcusolver);
    assert(err == hipSuccess);
    return err;
}


/*
 * MATRIX ROUTINES
 */

/* C = alpha op(A) op(B) + beta C
 * m: #rows of op(A)
 * n: #cols of op(B)
 * k: #cols of op(A) = #rows of op(B)
 */
template<typename Number>
int cu_multMat(
    hipblasHandle_t hcublas, size_t m, size_t n, size_t k, const Number* alpha,
    const Number* d_A, const Number* d_B, const Number* beta, Number* d_C,
    int dagger
) {
    assert(alpha);
    assert(d_A);
    assert(d_B);
    assert(beta);
    assert(dagger == 0 || dagger == 1 || dagger == 2);

    hipblasStatus_t err;
    if(dagger == 0) {
        err = cublasGemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, d_A, m, d_B, k,
            beta, d_C, m
        );
    } else if(dagger == 1) {
        err = cublasGemm(
            hcublas, HIPBLAS_OP_C, HIPBLAS_OP_N, m, n, k, alpha, d_A, k, d_B, k,
            beta, d_C, m
        );
    } else if(dagger == 2) {
        err = cublasGemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_C, m, n, k, alpha, d_A, m, d_B, n,
            beta, d_C, m
        );
    } else {
        std::fprintf(stderr, "expected dagger in [0, 1, 2], got %d\n", dagger);
        std::exit(EXIT_FAILURE);
    }
    assert(err == HIPBLAS_STATUS_SUCCESS);
    return err;
}

extern "C" int cu_CmultMat(
    hipblasHandle_t hcublas, size_t m, size_t n, size_t k,
    const hipComplex* alpha, const hipComplex* d_A, const hipComplex* d_B,
    const hipComplex* beta, hipComplex* d_C, int dagger
) {
    return cu_multMat(hcublas, m, n, k, alpha, d_A, d_B, beta, d_C, dagger);
}

extern "C" int cu_ZmultMat(
    hipblasHandle_t hcublas, size_t m, size_t n, size_t k,
    const hipDoubleComplex* alpha, const hipDoubleComplex* d_A,
    const hipDoubleComplex* d_B, const hipDoubleComplex* beta,
    hipDoubleComplex* d_C, int dagger
) {
    return cu_multMat(hcublas, m, n, k, alpha, d_A, d_B, beta, d_C, dagger);
}


extern "C" int cu_Cinverse(
    hipblasHandle_t hcublas, hipsolverHandle_t hcusolver, void* d_A,
    void* d_Ainv, size_t n
) {
    assert(hcusolver);
    assert(d_A);
    assert(d_Ainv);

    // compute buffer size and prep . memory
    hipComplex* pdA = (hipComplex*)d_A;
    int lwork;
    hipsolverStatus_t cusolverStatus =
        hipsolverDnCgetrf_bufferSize(hcusolver, n, n, pdA, n, &lwork);
    assert(cusolverStatus == HIPSOLVER_STATUS_SUCCESS);

    // prepare memory on the device
    hipComplex* d_LU;
    hipError_t cudaStatus =
        hipMalloc((void**)&d_LU, n * n * sizeof(hipComplex));
    assert(cudaStatus == hipSuccess);

    int* d_pivot;
    cudaStatus = hipMalloc((void**)&d_pivot, n * sizeof(int));
    assert(cudaStatus == hipSuccess);
    int* d_info;
    cudaStatus = hipMalloc((void**)&d_info, sizeof(int));
    assert(cudaStatus == hipSuccess);
    // copy d_LU <- pdA
    hipblasStatus_t hipblasStatus_t = hipblasCcopy(hcublas, n * n, pdA, 1, d_LU, 1);
    assert(hipblasStatus_t == HIPBLAS_STATUS_SUCCESS);

    hipComplex* d_work;
    cudaStatus = hipMalloc((void**)&d_work, lwork * sizeof(hipComplex));
    assert(cudaStatus == hipSuccess);

    // LU factorization of d_A , with partial pivoting and row
    // interchanges ; row i is interchanged with row d_pivot ( i );
    cusolverStatus =
        hipsolverDnCgetrf(hcusolver, n, n, d_LU, n, d_work, d_pivot, d_info);

    // use the LU factorization to solve the system d_LU * x = d_Ainv ;
    // the solution overwrites d_Ainv
    hipComplex* pdAinv = (hipComplex*)d_Ainv;
    cusolverStatus = hipsolverDnCgetrs(
        hcusolver, HIPBLAS_OP_N, n, n, d_LU, n, d_pivot, pdAinv, n, d_info
    );
    assert(cusolverStatus == HIPSOLVER_STATUS_SUCCESS);

    int info_gpu;
    // d_info -> info_gpu
    cudaStatus =
        hipMemcpy(&info_gpu, d_info, sizeof(int), hipMemcpyDeviceToHost);
    assert(cudaStatus == hipSuccess);

    cudaStatus = hipFree(d_pivot);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_info);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_work);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_LU);
    assert(cudaStatus == hipSuccess);

    return cudaStatus;
}

extern "C" int cu_Zinverse(
    hipblasHandle_t hcublas, hipsolverHandle_t hcusolver, void* d_A,
    void* d_Ainv, size_t n
) {
    assert(hcusolver);
    assert(d_A);
    assert(d_Ainv);

    // compute buffer size and prep . memory
    hipDoubleComplex* pdA = (hipDoubleComplex*)d_A;
    int lwork;
    hipsolverStatus_t cusolverStatus =
        hipsolverDnZgetrf_bufferSize(hcusolver, n, n, pdA, n, &lwork);
    assert(cusolverStatus == HIPSOLVER_STATUS_SUCCESS);

    // prepare memory on the device
    hipDoubleComplex* d_LU;
    hipError_t cudaStatus =
        hipMalloc((void**)&d_LU, n * n * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);

    int* d_pivot;
    cudaStatus = hipMalloc((void**)&d_pivot, n * sizeof(int));
    assert(cudaStatus == hipSuccess);
    int* d_info;
    cudaStatus = hipMalloc((void**)&d_info, sizeof(int));
    assert(cudaStatus == hipSuccess);
    // copy d_LU <- pdA
    hipblasStatus_t hipblasStatus_t = hipblasZcopy(hcublas, n * n, pdA, 1, d_LU, 1);
    assert(hipblasStatus_t == HIPBLAS_STATUS_SUCCESS);

    hipDoubleComplex* d_work;
    cudaStatus = hipMalloc((void**)&d_work, lwork * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);

    // LU factorization of d_A , with partial pivoting and row
    // interchanges ; row i is interchanged with row d_pivot ( i );
    cusolverStatus =
        hipsolverDnZgetrf(hcusolver, n, n, d_LU, n, d_work, d_pivot, d_info);

    // use the LU factorization to solve the system d_LU * x = d_Ainv ;
    // the solution overwrites d_Ainv
    hipDoubleComplex* pdAinv = (hipDoubleComplex*)d_Ainv;
    cusolverStatus = hipsolverDnZgetrs(
        hcusolver, HIPBLAS_OP_N, n, n, d_LU, n, d_pivot, pdAinv, n, d_info
    );
    assert(cusolverStatus == HIPSOLVER_STATUS_SUCCESS);

    int info_gpu;
    // d_info -> info_gpu
    cudaStatus =
        hipMemcpy(&info_gpu, d_info, sizeof(int), hipMemcpyDeviceToHost);
    assert(cudaStatus == hipSuccess);

    cudaStatus = hipFree(d_pivot);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_info);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_work);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_LU);
    assert(cudaStatus == hipSuccess);

    return cudaStatus;
}

template<typename Number>
int cu_kernelsum(
    Number* d_C, Number* alpha, Number* d_A, Number* beta, Number* d_B,
    size_t size
) {
    auto num_blocks = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;

    addKernel<<<num_blocks, BLOCK_SIZE>>>(d_C, *alpha, d_A, *beta, d_B, size);
    assert(hipGetLastError() == hipSuccess);

    return 0;
}


extern "C" int cu_Ckernelsum(
    hipComplex* d_C, hipComplex* alpha, hipComplex* d_A, hipComplex* beta,
    hipComplex* d_B, size_t size
) {
    return cu_kernelsum(d_C, alpha, d_A, beta, d_B, size);
}

extern "C" int cu_Zkernelsum(
    hipDoubleComplex* d_C, hipDoubleComplex* alpha, hipDoubleComplex* d_A,
    hipDoubleComplex* beta, hipDoubleComplex* d_B, size_t size
) {
    return cu_kernelsum(d_C, alpha, d_A, beta, d_B, size);
}


template<typename Number>
int cu_matsum(
    hipblasHandle_t hcublas, size_t m, size_t n, const Number* alpha,
    const Number* d_A, const Number* beta, const Number* d_B, Number* d_C,
    int dagger
) {
    hipblasStatus_t err;
    if(dagger == 0) {
        err = cublasGeam(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, alpha, d_A, m, beta, d_B,
            m, d_C, m
        );
    } else if(dagger == 1) {
        err = cublasGeam(
            hcublas, HIPBLAS_OP_C, HIPBLAS_OP_N, m, n, alpha, d_A, n, beta, d_B,
            m, d_C, m
        );
    } else if(dagger == 2) {
        err = cublasGeam(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_C, m, n, alpha, d_A, m, beta, d_B,
            n, d_C, m
        );
    } else {
        std::fprintf(stderr, "expected dagger in [0, 1, 2], got %d\n", dagger);
        std::exit(EXIT_FAILURE);
    }

    return err;
}

extern "C" int cu_Cmatsum(
    hipblasHandle_t hcublas, size_t m, size_t n, const hipComplex* alpha,
    const hipComplex* d_A, const hipComplex* beta, const hipComplex* d_B,
    hipComplex* d_C, int dagger
) {
    return cu_matsum(hcublas, m, n, alpha, d_A, beta, d_B, d_C, dagger);
}

extern "C" int cu_Zmatsum(
    hipblasHandle_t hcublas, size_t m, size_t n, const hipDoubleComplex* alpha,
    const hipDoubleComplex* d_A, const hipDoubleComplex* beta,
    const hipDoubleComplex* d_B, hipDoubleComplex* d_C, int dagger
) {
    return cu_matsum(hcublas, m, n, alpha, d_A, beta, d_B, d_C, dagger);
}

extern "C" int cu_Cinitmat(hipComplex* d_A, size_t nrow) {
    assert(d_A);
    auto size = nrow * nrow;
    auto num_blocks = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;

    initKernel<<<num_blocks, BLOCK_SIZE>>>(d_A, nrow);
    assert(hipGetLastError() == hipSuccess);

    return 0;
}

extern "C" int cu_Zinitmat(hipDoubleComplex* d_A, size_t nrow) {
    assert(d_A);
    auto size = nrow * nrow;
    auto num_blocks = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;

    initKernel<<<num_blocks, BLOCK_SIZE>>>(d_A, nrow);
    assert(hipGetLastError() == hipSuccess);

    return 0;
}


extern "C" float cu_Ctrace(
    hipblasHandle_t hcublas, void* d_A, size_t nrow, void* h_mask,
    int mask_present
) {
    hipComplex* pdA = (hipComplex*)d_A;
    auto size = nrow * nrow;
    auto num_blocks = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    float* d_work;
    hipError_t cudaStatus = hipMalloc((void**)&d_work, nrow * sizeof(float));
    float* d_iden;
    cudaStatus = hipMalloc((void**)&d_iden, nrow * sizeof(float));
    assert(cudaStatus == hipSuccess);
    bool* d_mask;
    cudaStatus = hipMalloc((void**)&d_mask, nrow * sizeof(bool));
    assert(cudaStatus == hipSuccess);
    if(h_mask) {
        cudaStatus = hipMemcpy(
            d_mask, h_mask, nrow * sizeof(bool), hipMemcpyHostToDevice
        );
        assert(cudaStatus == hipSuccess);
    }

    SinitKernel<<<num_blocks, BLOCK_SIZE>>>(d_iden, nrow);
    CtraceKernel<<<num_blocks, BLOCK_SIZE>>>(
        pdA, nrow, d_work, d_mask, mask_present
    );

    float result;
    hipblasStatus_t err =
        hipblasSdot(hcublas, nrow, d_iden, 1, d_work, 1, &result);
    assert(err == HIPBLAS_STATUS_SUCCESS);

    cudaStatus = hipFree(d_work);
    cudaStatus = hipFree(d_iden);
    cudaStatus = hipFree(d_mask);

    return result;
}

extern "C" double cu_Ztrace(
    hipblasHandle_t hcublas, void* d_A, size_t nrow, void* h_mask,
    int mask_present
) {
    assert(d_A);
    assert(h_mask || mask_present == 0);
    assert(mask_present == 0 || mask_present == 1);

    hipDoubleComplex* pdA = (hipDoubleComplex*)d_A;
    auto size = nrow * nrow;
    auto num_blocks = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    double* d_work;
    hipError_t cudaStatus = hipMalloc((void**)&d_work, nrow * sizeof(double));
    assert(cudaStatus == hipSuccess);
    double* d_iden;
    cudaStatus = hipMalloc((void**)&d_iden, nrow * sizeof(double));
    assert(cudaStatus == hipSuccess);
    bool* d_mask;
    cudaStatus = hipMalloc((void**)&d_mask, nrow * sizeof(bool));
    assert(cudaStatus == hipSuccess);
    if(h_mask) {
        cudaStatus = hipMemcpy(
            d_mask, h_mask, nrow * sizeof(bool), hipMemcpyHostToDevice
        );
        assert(cudaStatus == hipSuccess);
    }

    DinitKernel<<<num_blocks, BLOCK_SIZE>>>(d_iden, nrow);
    assert(hipGetLastError() == hipSuccess);
    ZtraceKernel<<<num_blocks, BLOCK_SIZE>>>(
        pdA, nrow, d_work, d_mask, mask_present
    );
    assert(hipGetLastError() == hipSuccess);

    double result;
    hipblasStatus_t err =
        hipblasDdot(hcublas, nrow, d_iden, 1, d_work, 1, &result);
    assert(err == HIPBLAS_STATUS_SUCCESS);

    cudaStatus = hipFree(d_work);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_iden);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_mask);
    assert(cudaStatus == hipSuccess);

    return result;
}

extern "C" int cu_Cmatcopy(
    hipblasHandle_t hcublas, const hipComplex* d_A, hipComplex* d_B,
    size_t num_elements
) {
    assert(d_A);
    assert(d_B);
    assert(num_elements <= INT_MAX);

    auto err = hipblasCcopy(hcublas, num_elements, d_A, 1, d_B, 1);
    assert(err == HIPBLAS_STATUS_SUCCESS);
    return err;
}

extern "C" int cu_Zmatcopy(
    hipblasHandle_t hcublas, hipDoubleComplex* d_A, hipDoubleComplex* d_B,
    size_t num_elements
) {
    assert(d_A);
    assert(d_B);
    assert(num_elements <= INT_MAX);

    auto err = hipblasZcopy(hcublas, num_elements, d_A, 1, d_B, 1);
    assert(err == HIPBLAS_STATUS_SUCCESS);
    return err;
}

extern "C" int
cu_Casum(hipblasHandle_t hcublas, void* d_A, float* summ, size_t n) {
    hipComplex* pdA = (hipComplex*)d_A;

    auto err = hipblasScasum(hcublas, n, pdA, 1, summ);
    assert(err == HIPBLAS_STATUS_SUCCESS);
    return err;
}

extern "C" int
cu_Zasum(hipblasHandle_t hcublas, void* d_A, double* summ, size_t n) {
    hipDoubleComplex* pdA = (hipDoubleComplex*)d_A;

    auto err = hipblasDzasum(hcublas, n, pdA, 1, summ);
    assert(err == HIPBLAS_STATUS_SUCCESS);
    return err;
}

extern "C" int cu_Cdecimation(
    hipblasHandle_t hcublas, hipsolverHandle_t hcusolver, void* h_Go_out,
    void* h_Ao_in, void* h_Bo_in, void* h_Co_in, size_t n, int tf32, int* ncyc,
    hipComplex* one, hipComplex* mone, hipComplex* zero, float SGFACC
) {
    assert(h_Go_out);
    assert(h_Ao_in);
    assert(h_Bo_in);
    assert(h_Co_in);
    assert(tf32 == 0 || tf32 == 1);
    assert(ncyc);
    assert(one);
    assert(mone);
    assert(zero);
    assert(SGFACC > 0.0);

    auto num_elements = n * n;
    auto num_blocks = (num_elements + BLOCK_SIZE - 1) / BLOCK_SIZE;

    hipComplex* d_Ao;
    hipError_t cudaStatus =
        hipMalloc((void**)&d_Ao, num_elements * sizeof(hipComplex));
    assert(cudaStatus == hipSuccess);

    hipComplex* d_Bo;
    cudaStatus = hipMalloc((void**)&d_Bo, num_elements * sizeof(hipComplex));
    assert(cudaStatus == hipSuccess);

    hipComplex* d_Co;
    cudaStatus = hipMalloc((void**)&d_Co, num_elements * sizeof(hipComplex));
    assert(cudaStatus == hipSuccess);

    cudaStatus = hipMemcpy(
        d_Ao, h_Ao_in, n * n * sizeof(hipComplex), hipMemcpyHostToDevice
    );
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipMemcpy(
        d_Bo, h_Bo_in, n * n * sizeof(hipComplex), hipMemcpyHostToDevice
    );
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipMemcpy(
        d_Co, h_Co_in, n * n * sizeof(hipComplex), hipMemcpyHostToDevice
    );
    assert(cudaStatus == hipSuccess);

    hipblasStatus_t hipblasStatus_t =
        hipblasSetPointerMode(hcublas, HIPBLAS_POINTER_MODE_HOST);
    assert(hipblasStatus_t == hipSuccess);

    if(tf32 == 1) {
        hipblasStatus_t = hipblasSetMathMode(hcublas, HIPBLAS_TENSOR_OP_MATH);
        assert(hipblasStatus_t == hipSuccess);
    }

    hipComplex* d_Ao_s;
    cudaStatus = hipMalloc((void**)&d_Ao_s, n * n * sizeof(hipComplex));
    assert(cudaStatus == hipSuccess);
    hipComplex* d_C1;
    cudaStatus = hipMalloc((void**)&d_C1, n * n * sizeof(hipComplex));
    assert(cudaStatus == hipSuccess);
    hipComplex* d_Go;
    cudaStatus = hipMalloc((void**)&d_Go, n * n * sizeof(hipComplex));
    assert(cudaStatus == hipSuccess);
    int* d_pivot;
    cudaStatus = hipMalloc((void**)&d_pivot, n * sizeof(int));
    assert(cudaStatus == hipSuccess);
    hipComplex* d_T;
    cudaStatus = hipMalloc((void**)&d_T, n * n * sizeof(hipComplex));
    assert(cudaStatus == hipSuccess);
    hipComplex* d_Self;
    cudaStatus = hipMalloc((void**)&d_Self, n * n * sizeof(hipComplex));
    assert(cudaStatus == hipSuccess);
    int* d_info;
    cudaStatus = hipMalloc((void**)&d_info, sizeof(int));
    assert(cudaStatus == hipSuccess);

    int lwork;
    hipsolverStatus_t cusolverStatus =
        hipsolverDnCgetrf_bufferSize(hcusolver, n, n, d_Self, n, &lwork);
    assert(cusolverStatus == hipSuccess);
    hipComplex* d_work;
    cudaStatus = hipMalloc((void**)&d_work, lwork * sizeof(hipComplex));
    assert(cudaStatus == hipSuccess);

    hipblasStatus_t = hipblasCcopy(hcublas, n * n, d_Ao, 1, d_Ao_s, 1);
    assert(hipblasStatus_t == hipSuccess);

    bool okCo = false;
    for(int i1 = 1; i1 <= 300; i1++) {
        *ncyc = i1;

        initKernel<<<num_blocks, BLOCK_SIZE>>>(d_Go, n);

        hipblasStatus_t = hipblasCcopy(hcublas, n * n, d_Ao, 1, d_Self, 1);
        assert(hipblasStatus_t == hipSuccess);

        cusolverStatus = hipsolverDnCgetrf(
            hcusolver, n, n, d_Self, n, d_work, d_pivot, d_info
        );
        assert(cusolverStatus == hipSuccess);
        cusolverStatus = hipsolverDnCgetrs(
            hcusolver, HIPBLAS_OP_N, n, n, d_Self, n, d_pivot, d_Go, n, d_info
        );
        assert(cusolverStatus == hipSuccess);

        hipblasStatus_t = hipblasCgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Go, n, d_Co, n,
            zero, d_T, n
        );
        assert(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = hipblasCgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Co, n, d_T, n,
            zero, d_C1, n
        );
        assert(hipblasStatus_t == hipSuccess);

        float summ;
        hipblasStatus_t = hipblasScasum(hcublas, n * n, d_C1, 1, &summ);
        assert(hipblasStatus_t == hipSuccess);
        // printf("loop it= %d , summ= %f \n ", i1, summ);

        if(summ <= SGFACC) {
            if(okCo) {
                break;
            } else {
                okCo = true;
            }
        } else {
            okCo = false;
        }

        hipblasStatus_t = hipblasCgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Bo, n, d_T, n,
            zero, d_Self, n
        );
        assert(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = hipblasCaxpy(hcublas, n * n, mone, d_Self, 1, d_Ao_s, 1);
        assert(hipblasStatus_t == hipSuccess);
        hipblasStatus_t = hipblasCaxpy(hcublas, n * n, mone, d_Self, 1, d_Ao, 1);
        assert(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = hipblasCgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Go, n, d_Bo, n,
            zero, d_T, n
        );
        assert(hipblasStatus_t == hipSuccess);
        hipblasStatus_t = hipblasCgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, mone, d_Co, n, d_T, n,
            one, d_Ao, n
        );
        assert(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = hipblasCcopy(hcublas, n * n, d_C1, 1, d_Co, 1);
        assert(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = hipblasCgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Bo, n, d_T, n,
            zero, d_C1, n
        );
        assert(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = hipblasCcopy(hcublas, n * n, d_C1, 1, d_Bo, 1);
        assert(hipblasStatus_t == hipSuccess);
    }

    initKernel<<<num_blocks, BLOCK_SIZE>>>(d_Go, n);
    hipblasStatus_t = hipblasCcopy(hcublas, n * n, d_Ao_s, 1, d_Self, 1);
    assert(hipblasStatus_t == hipSuccess);
    cusolverStatus =
        hipsolverDnCgetrf(hcusolver, n, n, d_Self, n, d_work, d_pivot, d_info);
    assert(cusolverStatus == hipSuccess);
    cusolverStatus = hipsolverDnCgetrs(
        hcusolver, HIPBLAS_OP_N, n, n, d_Self, n, d_pivot, d_Go, n, d_info
    );
    assert(cusolverStatus == hipSuccess);

    cudaStatus = hipMemcpy(
        h_Go_out, d_Go, n * n * sizeof(hipComplex), hipMemcpyDeviceToHost
    );
    assert(cudaStatus == hipSuccess);

    cudaStatus = hipFree(d_pivot);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_info);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Ao);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Bo);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Co);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Go);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Ao_s);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_C1);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_T);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Self);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_work);
    assert(cudaStatus == hipSuccess);

    return cudaStatus;
}

extern "C" int cu_Zdecimation(
    hipblasHandle_t hcublas, hipsolverHandle_t hcusolver, void* h_Go_out,
    void* h_Ao_in, void* h_Bo_in, void* h_Co_in, size_t n, int tf32, int* ncyc,
    hipDoubleComplex* one, hipDoubleComplex* mone, hipDoubleComplex* zero,
    double SGFACC
) {
    assert(h_Go_out);
    assert(h_Ao_in);
    assert(h_Bo_in);
    assert(h_Co_in);
    assert(tf32 == 0 || tf32 == 1);
    assert(ncyc);
    assert(one);
    assert(mone);
    assert(zero);
    assert(SGFACC > 0.0);

    auto num_elements = n * n;
    auto num_blocks = (num_elements + BLOCK_SIZE - 1) / BLOCK_SIZE;

    hipDoubleComplex* d_Ao;
    hipError_t cudaStatus =
        hipMalloc((void**)&d_Ao, num_elements * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);

    hipDoubleComplex* d_Bo;
    cudaStatus =
        hipMalloc((void**)&d_Bo, num_elements * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);

    hipDoubleComplex* d_Co;
    cudaStatus =
        hipMalloc((void**)&d_Co, num_elements * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);

    cudaStatus = hipMemcpy(
        d_Ao, h_Ao_in, n * n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice
    );
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipMemcpy(
        d_Bo, h_Bo_in, n * n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice
    );
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipMemcpy(
        d_Co, h_Co_in, n * n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice
    );
    assert(cudaStatus == hipSuccess);

    hipblasStatus_t hipblasStatus_t =
        hipblasSetPointerMode(hcublas, HIPBLAS_POINTER_MODE_HOST);
    assert(hipblasStatus_t == hipSuccess);

    if(tf32 == 1) {
        hipblasStatus_t = hipblasSetMathMode(hcublas, HIPBLAS_TENSOR_OP_MATH);
        assert(hipblasStatus_t == hipSuccess);
    }

    hipDoubleComplex* d_Ao_s;
    cudaStatus = hipMalloc((void**)&d_Ao_s, n * n * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);
    hipDoubleComplex* d_C1;
    cudaStatus = hipMalloc((void**)&d_C1, n * n * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);
    hipDoubleComplex* d_Go;
    cudaStatus = hipMalloc((void**)&d_Go, n * n * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);
    int* d_pivot;
    cudaStatus = hipMalloc((void**)&d_pivot, n * sizeof(int));
    assert(cudaStatus == hipSuccess);
    hipDoubleComplex* d_T;
    cudaStatus = hipMalloc((void**)&d_T, n * n * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);
    hipDoubleComplex* d_Self;
    cudaStatus = hipMalloc((void**)&d_Self, n * n * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);
    int* d_info;
    cudaStatus = hipMalloc((void**)&d_info, sizeof(int));
    assert(cudaStatus == hipSuccess);

    int lwork;
    hipsolverStatus_t cusolverStatus =
        hipsolverDnZgetrf_bufferSize(hcusolver, n, n, d_Self, n, &lwork);
    assert(cusolverStatus == hipSuccess);
    hipDoubleComplex* d_work;
    cudaStatus = hipMalloc((void**)&d_work, lwork * sizeof(hipDoubleComplex));
    assert(cudaStatus == hipSuccess);

    hipblasStatus_t = hipblasZcopy(hcublas, n * n, d_Ao, 1, d_Ao_s, 1);
    assert(hipblasStatus_t == hipSuccess);

    bool okCo = false;
    for(int i1 = 1; i1 <= 300; i1++) {
        *ncyc = i1;

        initKernel<<<num_blocks, BLOCK_SIZE>>>(d_Go, n);

        hipblasStatus_t = hipblasZcopy(hcublas, n * n, d_Ao, 1, d_Self, 1);
        assert(hipblasStatus_t == hipSuccess);

        cusolverStatus = hipsolverDnZgetrf(
            hcusolver, n, n, d_Self, n, d_work, d_pivot, d_info
        );
        assert(cusolverStatus == hipSuccess);
        cusolverStatus = hipsolverDnZgetrs(
            hcusolver, HIPBLAS_OP_N, n, n, d_Self, n, d_pivot, d_Go, n, d_info
        );
        assert(cusolverStatus == hipSuccess);

        hipblasStatus_t = hipblasZgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Go, n, d_Co, n,
            zero, d_T, n
        );
        assert(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = hipblasZgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Co, n, d_T, n,
            zero, d_C1, n
        );
        assert(hipblasStatus_t == hipSuccess);

        double summ;
        hipblasStatus_t = hipblasDzasum(hcublas, n * n, d_C1, 1, &summ);
        assert(hipblasStatus_t == hipSuccess);
        //printf("loop it= %d , summ= %f \n ", i1, summ);

        if(summ <= SGFACC) {
            if(okCo) {
                break;
            } else {
                okCo = true;
            }
        } else {
            okCo = false;
        }

        hipblasStatus_t = hipblasZgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Bo, n, d_T, n,
            zero, d_Self, n
        );
        assert(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = hipblasZaxpy(hcublas, n * n, mone, d_Self, 1, d_Ao_s, 1);
        assert(hipblasStatus_t == hipSuccess);
        hipblasStatus_t = hipblasZaxpy(hcublas, n * n, mone, d_Self, 1, d_Ao, 1);
        assert(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = hipblasZgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Go, n, d_Bo, n,
            zero, d_T, n
        );
        assert(hipblasStatus_t == hipSuccess);
        hipblasStatus_t = hipblasZgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, mone, d_Co, n, d_T, n,
            one, d_Ao, n
        );
        assert(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = hipblasZcopy(hcublas, n * n, d_C1, 1, d_Co, 1);
        assert(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = hipblasZgemm(
            hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Bo, n, d_T, n,
            zero, d_C1, n
        );
        assert(hipblasStatus_t == hipSuccess);

        hipblasStatus_t = hipblasZcopy(hcublas, n * n, d_C1, 1, d_Bo, 1);
        assert(hipblasStatus_t == hipSuccess);
    }

    initKernel<<<num_blocks, BLOCK_SIZE>>>(d_Go, n);
    hipblasStatus_t = hipblasZcopy(hcublas, n * n, d_Ao_s, 1, d_Self, 1);
    assert(hipblasStatus_t == hipSuccess);
    cusolverStatus =
        hipsolverDnZgetrf(hcusolver, n, n, d_Self, n, d_work, d_pivot, d_info);
    assert(cusolverStatus == hipSuccess);
    cusolverStatus = hipsolverDnZgetrs(
        hcusolver, HIPBLAS_OP_N, n, n, d_Self, n, d_pivot, d_Go, n, d_info
    );
    assert(cusolverStatus == hipSuccess);

    cudaStatus = hipMemcpy(
        h_Go_out, d_Go, n * n * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost
    );
    assert(cudaStatus == hipSuccess);

    cudaStatus = hipFree(d_pivot);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_info);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Ao);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Bo);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Co);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Go);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Ao_s);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_C1);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_T);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_Self);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipFree(d_work);
    assert(cudaStatus == hipSuccess);

    return cudaStatus;
}

extern "C" int cu_meminfo(size_t* freemem, size_t* totalmem) {
    hipError_t cudaStatus;
    cudaStatus = hipDeviceSynchronize();
    cudaStatus = hipMemGetInfo(freemem, totalmem);
    assert(cudaStatus == hipSuccess);
    return cudaStatus;
}
