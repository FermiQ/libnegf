/*!!--------------------------------------------------------------------------!
 *!! libNEGF: a general library for Non-Equilibrium Greens functions.         !
 *!! Copyright (C) 2012 - 2026                                                !
 *!!                                                                          !
 *!! This file is part of libNEGF: a library for                              !
 *!! Non Equilibrium Green's Function calculation                             !
 *!!                                                                          !
 *!! Developers: Alessandro Pecchia, Daniele Soccodato                        !
 *!! Former Contributors: Gabriele Penazzi, Luca Latessa, Aldo Di Carlo       !
 *!!                                                                          !
 *!! libNEGF is free software: you can redistribute it and/or modify          !
 *!! it under the terms of the GNU Lesse General Public License as published  !
 *!! by the Free Software Foundation, either version 3 of the License, or     !
 *!! (at your option) any later version.                                      !
 *!!                                                                          !
 *!!  You should have received a copy of the GNU Lesser General Public        !
 *!!  License along with libNEGF.  If not, see                                !
 *!!  <http://www.gnu.org/licenses/>.                                         !
 *!!--------------------------------------------------------------------------!
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include "hipblas.h"
#include "hipsolver.h"

#define BLOCK_SIZE 1024
#define TILE_DIM 32
#define BLOCK_ROWS 8

__global__ void CaddKernel(hipComplex* c, const hipComplex alpha, const hipComplex* a, const hipComplex beta, const hipComplex* b, int size)
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < size) {
      c[i].x = (alpha.x * a[i].x - alpha.y * a[i].y) + (beta.x * b[i].x - beta.y * b[i].y);
      c[i].y = (alpha.x * a[i].y + alpha.y * a[i].x) + (beta.x * b[i].y + beta.y * b[i].x);
   }
}

__global__ void ZaddKernel(hipDoubleComplex* c, const hipDoubleComplex alpha, const hipDoubleComplex* a, const hipDoubleComplex beta, const hipDoubleComplex* b, int size)
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < size) {
      c[i].x = (alpha.x * a[i].x - alpha.y * a[i].y) + (beta.x * b[i].x - beta.y * b[i].y);
      c[i].y = (alpha.x * a[i].y + alpha.y * a[i].x) + (beta.x * b[i].y + beta.y * b[i].x);
   }
}

/*
__global__ void hermitian(cuComplex *odata, const cuComplex *idata)
{
  __shared__ cuComplex tile[TILE_DIM][TILE_DIM];

  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
  {
     odata[(y+j)*width + x].x = tile[threadIdx.x][threadIdx.y + j].x;
     odata[(y+j)*width + x].y = -tile[threadIdx.x][threadIdx.y + j].y;
  }
}
*/
__global__ void CinitKernel(hipComplex *a, int nrow) {

    int size;
    int i = blockDim.x*blockIdx.x + threadIdx.x;

    size = nrow*nrow;
    if(i < size) {
          if(i%(nrow+1) == 0){
              a[i].x = 1.0;
              a[i].y = 0.0;
	     }
          else{
              a[i].x = 0.0;
              a[i].y = 0.0;
	   }
    }
}

__global__ void ZinitKernel(hipDoubleComplex *a, int nrow) {

    int size;
    int i = blockDim.x*blockIdx.x + threadIdx.x;

    size = nrow*nrow;
    if(i < size) {
          if(i%(nrow+1) == 0){
              a[i].x = 1.0;
              a[i].y = 0.0;
	     }
          else{
              a[i].x = 0.0;
              a[i].y = 0.0;
	   }
    }
}

__global__ void DinitKernel(double *a, int nrow) {

    int i = blockDim.x*blockIdx.x + threadIdx.x;

    if(i < nrow) {
              a[i] = 1.0;
    }
}

__global__ void SinitKernel(float *a, int nrow) {

    int i = blockDim.x*blockIdx.x + threadIdx.x;

    if(i < nrow) {
              a[i] = 1.0;
    }
}

__global__ void CtraceKernel(hipComplex *a, int nrow, float *trace, bool *mask, int mask_present) {

    int size;
    int i = blockDim.x*blockIdx.x + threadIdx.x;

    size = nrow*nrow;
    if (mask_present == 0){
       if(i < size) {
          if(i%(nrow+1) == 0){
             trace[i%nrow] = a[i].x;
             }
       }
    }
    if(mask_present == 1){
       if(i < size) {
          if(i%(nrow+1) == 0){
	     if(mask[i%nrow]){
                trace[i%nrow] = a[i].x;
	     }
	  }
       }
    }
}

__global__ void ZtraceKernel(hipDoubleComplex *a, int nrow, double  *trace, bool *mask, int mask_present) {

    int size;
    int i = blockDim.x*blockIdx.x + threadIdx.x;

    size = nrow*nrow;
    if (mask_present == 0){
       if(i < size) {
          if(i%(nrow+1) == 0){
             trace[i%nrow] = a[i].x;
             }
       }
    }
    if(mask_present == 1){
       if(i < size) {
          if(i%(nrow+1) == 0){
	     if(mask[i%nrow]){
                trace[i%nrow] = a[i].x;
	     }
	  }
       }
    }
}

/*~-~-~-~-~-~-~-~-~-~-~-~-~-~ DATA MOVEMENT  ROUTINES -~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-*/

extern "C" int cu_createMat(void **d_A, int bytecount)
{
  hipError_t err;
  err = hipMalloc(d_A, bytecount);
  //printf("GPU Address: %p \n",*d_A);
  return err;
}

extern "C" int cu_copyMatH2D(void *h_A, void *d_A, int bytecount)
{
  hipError_t err;
  //printf("copy %p to %p\n",h_A,d_A);
  err = hipMemcpy(d_A, h_A, bytecount, hipMemcpyHostToDevice);
  return err;
}

extern "C" int cu_copyMatD2H(void *h_A, void *d_A, int bytecount)
{
  hipError_t err;
  //printf("copy %p to %p\n",d_A,h_A);
  err = hipMemcpy(h_A, d_A, bytecount, hipMemcpyDeviceToHost);
  return err;
}

extern "C" int cu_deleteMat(void *d_A)
{
  hipError_t err;
  //printf("add_free: %p",d_A);
  err = hipFree(d_A);
  return err;
}

/*~-~-~-~-~-~-~-~-~-~-~-~-~-~ INIT/FINAL ROUTINES -~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-*/

extern "C" int cu_cudaGetDeviceCount(int *count)
{
  hipError_t err;

  err = hipGetDeviceCount(count);
  return err;
}


extern "C" int cu_cudaGetDeviceProperties(int device)
{
  hipError_t err;
  hipDeviceProp_t prop;

  err = hipGetDeviceProperties(&prop, device);

  printf(" Found GPU: Device Name: %s\n",prop.name);
  printf(" TotalMemory: %lu\n",(unsigned long) prop.totalGlobalMem);
  printf(" Shared per block: %lu\n",(unsigned long) prop.sharedMemPerBlock);
  return err;
}




extern "C" int cu_cublasInit(hipblasHandle_t *hcublas)
{
  hipblasStatus_t err;
  err = hipblasCreate(hcublas);
  if (err != 0){
    printf("cublas create error: %d\n",err);
  }
  //printf("hcublas Addr: %p \n",*hcublas);
  return err;
}

extern "C" int cu_cublasFinalize(hipblasHandle_t hcublas)
{
  hipblasStatus_t err;
  err = hipblasDestroy(hcublas);
  return err;
}

extern "C" int cu_cusolverInit(hipsolverHandle_t *hcusolver)
{
  hipsolverStatus_t err;
  err = hipsolverDnCreate(hcusolver);
  if (err != 0){
    printf("cusolver create error: %d\n",err);
  }
  //printf("hcusolver Addr: %p \n",*hcusolver);
  return err;
}

extern "C" int cu_cusolverFinalize(hipsolverHandle_t hcusolver)
{
  hipsolverStatus_t err;
  err = hipsolverDnDestroy(hcusolver);
  return err;
}

/*~-~-~-~-~-~-~-~-~-~-~-~-~-~ MATRIX ROUTINES -~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-*/
/* C = alpha op(A) op(B) + beta C
 * m: #rows of op(A)
 * n: #cols of op(B)
 * k: #cols of op(A) = #rows of op(B)
 */
extern "C" int cu_CmultMat(hipblasHandle_t hcublas, int m, int n, int k, hipComplex *alpha, void *d_A, void *d_B, hipComplex *beta, void *d_C, int dagger)
{
  hipComplex *pdA, *pdB, *pdC;

  //printf("A: %p B: %p C: %p\n",d_A,d_B,d_C);
  pdA=(hipComplex *) d_A;
  pdB=(hipComplex *) d_B;
  pdC=(hipComplex *) d_C;
  hipblasStatus_t err;
  if (dagger == 0){
     err = hipblasCgemm(hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, pdA, m, pdB, k, beta, pdC, m);
  }
  if (dagger == 1){
     err = hipblasCgemm(hcublas, HIPBLAS_OP_C, HIPBLAS_OP_N, m, n, k, alpha, pdA, k, pdB, k, beta, pdC, m);
  }
  if (dagger == 2){
     err = hipblasCgemm(hcublas, HIPBLAS_OP_N, HIPBLAS_OP_C, m, n, k, alpha, pdA, m, pdB, n, beta, pdC, m);
  }
  return err;
}

//  C = alpha op(A) op(B) + beta C
// op(A):  m x k
// op(B):  k x n
//     C:  m x n
extern "C" int cu_ZmultMat(hipblasHandle_t hcublas, int m, int n, int k, hipDoubleComplex *alpha, void *d_A, void *d_B, hipDoubleComplex *beta, void *d_C, int dagger)
{
  hipDoubleComplex *pdA, *pdB, *pdC;

  pdA=(hipDoubleComplex *) d_A;
  pdB=(hipDoubleComplex *) d_B;
  pdC=(hipDoubleComplex *) d_C;
  hipblasStatus_t err;
  if (dagger == 0){
     err = hipblasZgemm(hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, pdA, m, pdB, k, beta, pdC, m);
  }
  if (dagger == 1){
     err = hipblasZgemm(hcublas, HIPBLAS_OP_C, HIPBLAS_OP_N, m, n, k, alpha, pdA, k, pdB, k, beta, pdC, m);
  }
  if (dagger == 2){
     err = hipblasZgemm(hcublas, HIPBLAS_OP_N, HIPBLAS_OP_C, m, n, k, alpha, pdA, m, pdB, n, beta, pdC, m);
  }
  return err;
}

extern "C" int cu_Cinverse(hipblasHandle_t hcublas, hipsolverHandle_t hcusolver, void *d_A, void *d_Ainv, int N)
{
   hipError_t cudaStatus;
   hipsolverStatus_t cusolverStatus;
   hipblasStatus_t hipblasStatus_t;
   // declare arrays on the device
   hipComplex  *pdA , *pdAinv, *d_LU, *d_Work;

   pdA = (hipComplex *) d_A;
   pdAinv = (hipComplex *) d_Ainv;
   // coeff . matrix , rhs , workspace
   int *d_pivot , *d_info , Lwork ; // pivots , info , worksp . size
   int info_gpu = 0;

   // compute buffer size and prep . memory
   cusolverStatus = hipsolverDnCgetrf_bufferSize( hcusolver, N , N , pdA , N , &Lwork);
   // prepare memory on the device

   cudaStatus = hipMalloc(( void **)& d_LU, N*N*sizeof(hipComplex));
   cudaStatus = hipMalloc(( void **)& d_pivot , N*sizeof(int));
   cudaStatus = hipMalloc(( void **)& d_info , sizeof(int));
   // copy d_LU <- pdA
   hipblasStatus_t = hipblasCcopy(hcublas, N*N, pdA, 1, d_LU, 1);

   cudaStatus = hipMalloc(( void **)& d_Work , Lwork*sizeof(hipComplex));

   // LU factorization of d_A , with partial pivoting and row
   // interchanges ; row i is interchanged with row d_pivot ( i );
   cusolverStatus = hipsolverDnCgetrf(hcusolver, N, N, d_LU, N, d_Work, d_pivot, d_info);

   // use the LU factorization to solve the system d_LU * x = d_Ainv ;
   // the solution overwrites d_Ainv
   cusolverStatus = hipsolverDnCgetrs(hcusolver, HIPBLAS_OP_N, N, N, d_LU, N, d_pivot, pdAinv, N, d_info);

   cudaStatus = hipMemcpy(&info_gpu , d_info , sizeof(int), hipMemcpyDeviceToHost);
   // d_info -> info_gpu
   cudaStatus = hipFree(d_pivot);
   cudaStatus = hipFree(d_info);
   cudaStatus = hipFree(d_Work);
   cudaStatus = hipFree(d_LU);
   return cudaStatus;
}

extern "C" int cu_Zinverse(hipblasHandle_t hcublas, hipsolverHandle_t hcusolver, void *d_A, void *d_Ainv, int N)
{
   hipError_t cudaStatus;
   hipsolverStatus_t cusolverStatus;
   hipblasStatus_t hipblasStatus_t;
   // declare arrays on the device
   hipDoubleComplex  *pdA , *pdAinv, *d_LU, *d_Work;

   pdA = (hipDoubleComplex *) d_A;
   pdAinv = (hipDoubleComplex *) d_Ainv;
   // coeff . matrix , rhs , workspace
   int *d_pivot , *d_info , Lwork ; // pivots , info , worksp . size
   int info_gpu = 0;

   // compute buffer size and prep . memory
   cusolverStatus = hipsolverDnZgetrf_bufferSize( hcusolver, N , N , pdA , N , &Lwork);
   // prepare memory on the device

   cudaStatus = hipMalloc(( void **)& d_LU, N*N*sizeof(hipDoubleComplex));
   cudaStatus = hipMalloc(( void **)& d_pivot , N*sizeof(int));
   cudaStatus = hipMalloc(( void **)& d_info , sizeof(int));
   // copy d_LU <- pdA
   hipblasStatus_t = hipblasZcopy(hcublas, N*N, pdA, 1, d_LU, 1);

   cudaStatus = hipMalloc(( void **)& d_Work , Lwork*sizeof(hipDoubleComplex));

   // LU factorization of d_A , with partial pivoting and row
   // interchanges ; row i is interchanged with row d_pivot ( i );
   cusolverStatus = hipsolverDnZgetrf(hcusolver, N, N, d_LU, N, d_Work, d_pivot, d_info);

   // use the LU factorization to solve the system d_LU * x = d_Ainv ;
   // the solution overwrites d_Ainv
   cusolverStatus = hipsolverDnZgetrs(hcusolver, HIPBLAS_OP_N, N, N, d_LU, N, d_pivot, pdAinv, N, d_info);

   cudaStatus = hipMemcpy(&info_gpu , d_info , sizeof(int), hipMemcpyDeviceToHost);
   // d_info -> info_gpu
   cudaStatus = hipFree(d_pivot);
   cudaStatus = hipFree(d_info);
   cudaStatus = hipFree(d_Work);
   cudaStatus = hipFree(d_LU);
   return cudaStatus;
}

extern "C" int cu_Ckernelsum(void *d_C, hipComplex *alpha, void *d_A, hipComplex *beta, void *d_B, int size)
{
   int NumBlocks;
   hipComplex *pdA = (hipComplex *) d_A;
   hipComplex *pdB = (hipComplex *) d_B;
   hipComplex *pdC = (hipComplex *) d_C;

   NumBlocks = (size/BLOCK_SIZE)+1;

   CaddKernel<<<NumBlocks,BLOCK_SIZE>>>(pdC, *alpha, pdA, *beta, pdB, size);

   return 0;
}

extern "C" int cu_Zkernelsum(void *d_C, hipDoubleComplex *alpha, void *d_A, hipDoubleComplex *beta, void *d_B, int size)
{
   int NumBlocks;
   hipDoubleComplex *pdA = (hipDoubleComplex *) d_A;
   hipDoubleComplex *pdB = (hipDoubleComplex *) d_B;
   hipDoubleComplex *pdC = (hipDoubleComplex *) d_C;

   NumBlocks = (size/BLOCK_SIZE)+1;

   ZaddKernel<<<NumBlocks,BLOCK_SIZE>>>(pdC, *alpha, pdA, *beta, pdB, size);

   return 0;
}

extern "C" int cu_Cmatsum(hipblasHandle_t hcublas, int m, int n, hipComplex *alpha, void *d_A, hipComplex *beta, void *d_B, void *d_C, int dagger)
{
   //m number of rows of matrix op(A) and C
   //n number of columns of matrix op(B) and C
   hipComplex *pdA = (hipComplex *) d_A;
   hipComplex *pdB = (hipComplex *) d_B;
   hipComplex *pdC = (hipComplex *) d_C;

   hipblasStatus_t err;
   if (dagger == 0) {
      err = hipblasCgeam(hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, alpha, pdA, m, beta, pdB, m, pdC, m);
      }
   if (dagger == 1) {
      err = hipblasCgeam(hcublas, HIPBLAS_OP_C, HIPBLAS_OP_N, m, n, alpha, pdA, n, beta, pdB, m, pdC, m);
      }
   if (dagger == 2) {
      err = hipblasCgeam(hcublas, HIPBLAS_OP_N, HIPBLAS_OP_C, m, n, alpha, pdA, m, beta, pdB, n, pdC, m);
      }
   return err;
}

extern "C" int cu_Zmatsum(hipblasHandle_t hcublas, int m, int n, hipDoubleComplex *alpha, void *d_A, hipDoubleComplex *beta, void *d_B, void *d_C, int dagger)
{
   //m number of rows of matrix op(A) and C
   //n number of columns of matrix op(B) and C
   hipDoubleComplex *pdA = (hipDoubleComplex *) d_A;
   hipDoubleComplex *pdB = (hipDoubleComplex *) d_B;
   hipDoubleComplex *pdC = (hipDoubleComplex *) d_C;

   hipblasStatus_t err;
   if (dagger == 0) {
      err = hipblasZgeam(hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, alpha, pdA, m, beta, pdB, m, pdC, m);
      }
   if (dagger == 1) {
      err = hipblasZgeam(hcublas, HIPBLAS_OP_C, HIPBLAS_OP_N, m, n, alpha, pdA, n, beta, pdB, m, pdC, m);
      }
   if (dagger == 2) {
      err = hipblasZgeam(hcublas, HIPBLAS_OP_N, HIPBLAS_OP_C, m, n, alpha, pdA, m, beta, pdB, n, pdC, m);
      }
   return err;
}

extern "C" int cu_Cinitmat( void *d_A, int nrow)
{
   int NumBlocks;
   int size = nrow*nrow;
   hipComplex *pdA = (hipComplex *) d_A;

   NumBlocks = (size/BLOCK_SIZE)+1;

   CinitKernel<<<NumBlocks,BLOCK_SIZE>>>(pdA, nrow);

   return 0;
}

extern "C" int cu_Zinitmat( void *d_A, int nrow)
{
   int NumBlocks;
   int size = nrow*nrow;
   hipDoubleComplex *pdA = (hipDoubleComplex *) d_A;

   NumBlocks = (size/BLOCK_SIZE)+1;

   ZinitKernel<<<NumBlocks,BLOCK_SIZE>>>(pdA, nrow);

   return 0;
}

extern "C" float cu_Ctrace(hipblasHandle_t hcublas, void *d_A, int nrow, void *h_mask, int mask_present)
{
   hipError_t cudaStatus;
   hipblasStatus_t err;

   int NumBlocks;
   float result;
   int size = nrow*nrow;

   float *d_work, *d_iden;
   hipComplex *pdA = (hipComplex *) d_A;
   bool *phmask = (bool *) h_mask;
   bool *d_mask;

   NumBlocks = (size/BLOCK_SIZE)+1;

   cudaStatus = hipMalloc(( void **)& d_work, nrow*sizeof(float));
   cudaStatus = hipMalloc(( void **)& d_iden, nrow*sizeof(float));
   cudaStatus = hipMalloc(( void **)& d_mask, nrow*sizeof(bool));
   cudaStatus = hipMemcpy(d_mask, phmask, nrow*sizeof(bool), hipMemcpyHostToDevice);

   SinitKernel<<<NumBlocks,BLOCK_SIZE>>>(d_iden, nrow);
   CtraceKernel<<<NumBlocks,BLOCK_SIZE>>>(pdA, nrow, d_work, d_mask, mask_present);

   err =  hipblasSdot (hcublas, nrow, d_iden, 1, d_work, 1, &result);

   cudaStatus = hipFree(d_work);
   cudaStatus = hipFree(d_iden);
   cudaStatus = hipFree(d_mask);

   return result;
}

extern "C" double cu_Ztrace(hipblasHandle_t hcublas, void *d_A, int nrow, void *h_mask, int mask_present)
{
   hipError_t cudaStatus;
   hipblasStatus_t err;

   int NumBlocks;
   double result;
   int size = nrow*nrow;

   double *d_work, *d_iden;
   hipDoubleComplex *pdA = (hipDoubleComplex *) d_A;
   bool *phmask = (bool *) h_mask;
   bool *d_mask;

   NumBlocks = (size/BLOCK_SIZE)+1;

   cudaStatus = hipMalloc(( void **)& d_work, nrow*sizeof(double));
   cudaStatus = hipMalloc(( void **)& d_iden, nrow*sizeof(double));
   cudaStatus = hipMalloc(( void **)& d_mask, nrow*sizeof(bool));
   cudaStatus = hipMemcpy(d_mask, phmask, nrow*sizeof(bool), hipMemcpyHostToDevice);

   DinitKernel<<<NumBlocks,BLOCK_SIZE>>>(d_iden, nrow);
   ZtraceKernel<<<NumBlocks,BLOCK_SIZE>>>(pdA, nrow, d_work, d_mask, mask_present);

   err =  hipblasDdot(hcublas, nrow, d_iden, 1, d_work, 1, &result);

   cudaStatus = hipFree(d_work);
   cudaStatus = hipFree(d_iden);
   cudaStatus = hipFree(d_mask);

   return result;
}

extern "C" int cu_Cmatcopy(hipblasHandle_t hcublas,  void *d_A,  void *d_B, int N)
{
   hipComplex *pdA = (hipComplex *) d_A;
   hipComplex *pdB = (hipComplex *) d_B;

   hipblasStatus_t err;

   err = hipblasCcopy(hcublas, N*N, pdA, 1, pdB, 1);
   return err;
}

extern "C" int cu_Zmatcopy(hipblasHandle_t hcublas,  void *d_A,  void *d_B, int size)
{
   hipDoubleComplex *pdA = (hipDoubleComplex *) d_A;
   hipDoubleComplex *pdB = (hipDoubleComplex *) d_B;

   hipblasStatus_t err;

   err = hipblasZcopy(hcublas, size, pdA, 1, pdB, 1);
   return err;
}

extern "C" int cu_Casum(hipblasHandle_t hcublas,  void *d_A,  float *summ, int N)
{
   hipComplex *pdA = (hipComplex *) d_A;

   hipblasStatus_t err;

   err = hipblasScasum(hcublas, N, pdA, 1, summ);
   return err;
}

extern "C" int cu_Zasum(hipblasHandle_t hcublas,  void *d_A,  double *summ, int N)
{
   hipDoubleComplex *pdA = (hipDoubleComplex *) d_A;

   hipblasStatus_t err;

   err = hipblasDzasum(hcublas, N, pdA, 1, summ);
   return err;
}

extern "C" int cu_Cdecimation(hipblasHandle_t hcublas, hipsolverHandle_t hcusolver, void *h_Go_out, void *h_Ao_in, void *h_Bo_in, 
       void *h_Co_in, int n, int tf32, int *ncyc, hipComplex *one, hipComplex *mone, hipComplex *zero, float SGFACC)
{
   hipError_t cudaStatus;
   hipsolverStatus_t cusolverStatus;
   hipblasStatus_t hipblasStatus_t;

   float summ;
   bool okCo = false;
   hipComplex *phGo_out = (hipComplex *) h_Go_out;
   hipComplex *phAo_in = (hipComplex *) h_Ao_in;
   hipComplex *phBo_in = (hipComplex *) h_Bo_in;
   hipComplex *phCo_in = (hipComplex *) h_Co_in;

   hipComplex *d_Ao, *d_Bo, *d_Co, *d_Go, *d_Ao_s, *d_C1, *d_T, *d_Self, *d_work;
   int *d_pivot , *d_info , Lwork ; // pivots , info , worksp . size
   int i1, NumBlocks;

   NumBlocks = ((n*n)/BLOCK_SIZE)+1;

   cudaStatus = hipMalloc(( void **)& d_Ao, n*n*sizeof(hipComplex));
   cudaStatus = hipMalloc(( void **)& d_Bo, n*n*sizeof(hipComplex));
   cudaStatus = hipMalloc(( void **)& d_Co, n*n*sizeof(hipComplex));
   cudaStatus = hipMemcpy(d_Ao, phAo_in, n*n*sizeof(hipComplex), hipMemcpyHostToDevice);
   cudaStatus = hipMemcpy(d_Bo, phBo_in, n*n*sizeof(hipComplex), hipMemcpyHostToDevice);
   cudaStatus = hipMemcpy(d_Co, phCo_in, n*n*sizeof(hipComplex), hipMemcpyHostToDevice);


   hipblasStatus_t = hipblasSetPointerMode(hcublas, HIPBLAS_POINTER_MODE_HOST);
   if (tf32 == 1){
	   hipblasStatus_t = hipblasSetMathMode(hcublas, HIPBLAS_TENSOR_OP_MATH);
   }

   cudaStatus = hipMalloc(( void **)& d_Ao_s, n*n*sizeof(hipComplex));
   cudaStatus = hipMalloc(( void **)& d_C1, n*n*sizeof(hipComplex));
   cudaStatus = hipMalloc(( void **)& d_Go, n*n*sizeof(hipComplex));
   cudaStatus = hipMalloc(( void **)& d_pivot, n*sizeof(int));
   cudaStatus = hipMalloc(( void **)& d_T, n*n*sizeof(hipComplex));
   cudaStatus = hipMalloc(( void **)& d_Self, n*n*sizeof(hipComplex));
   cudaStatus = hipMalloc(( void **)& d_info, sizeof(int));

   cusolverStatus = hipsolverDnCgetrf_bufferSize(hcusolver, n, n, d_Self, n, &Lwork);
   cudaStatus = hipMalloc(( void **)& d_work, Lwork*sizeof(hipComplex));

   hipblasStatus_t = hipblasCcopy(hcublas, n*n, d_Ao, 1, d_Ao_s, 1);

   for(i1=1; i1<=300; i1++ ){
      *ncyc = i1;

      CinitKernel<<<NumBlocks,BLOCK_SIZE>>>(d_Go, n);

      hipblasStatus_t = hipblasCcopy(hcublas, n*n, d_Ao, 1, d_Self, 1);

      cusolverStatus = hipsolverDnCgetrf(hcusolver, n, n, d_Self, n, d_work, d_pivot, d_info);
      cusolverStatus = hipsolverDnCgetrs(hcusolver, HIPBLAS_OP_N, n, n, d_Self, n, d_pivot, d_Go, n, d_info);

      hipblasStatus_t = hipblasCgemm(hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Go, n, d_Co, n, zero, d_T, n);

      hipblasStatus_t = hipblasCgemm(hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Co, n, d_T, n, zero, d_C1, n);

      hipblasStatus_t = hipblasScasum(hcublas, n*n, d_C1, 1, &summ);
      //printf("loop it= %d , summ= %f \n ", i1, summ);

      if (summ <= SGFACC){
         if (okCo){
            break;
	 }
         else{
            okCo = true;
	 }
      }
      else{
         okCo = false;
      }

      hipblasStatus_t = hipblasCgemm(hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Bo, n, d_T, n, zero, d_Self, n);

      hipblasStatus_t = hipblasCaxpy(hcublas, n*n, mone, d_Self, 1, d_Ao_s, 1);
      hipblasStatus_t = hipblasCaxpy(hcublas, n*n, mone, d_Self, 1, d_Ao, 1);

      hipblasStatus_t = hipblasCgemm(hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Go, n, d_Bo, n, zero, d_T, n);
      hipblasStatus_t = hipblasCgemm(hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, mone, d_Co, n, d_T, n, one, d_Ao, n);

      hipblasStatus_t = hipblasCcopy(hcublas, n*n, d_C1, 1, d_Co, 1);

      hipblasStatus_t = hipblasCgemm(hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Bo, n, d_T, n, zero, d_C1, n);

      hipblasStatus_t = hipblasCcopy(hcublas, n*n, d_C1, 1, d_Bo, 1);
   }


   CinitKernel<<<NumBlocks,BLOCK_SIZE>>>(d_Go, n);
   hipblasStatus_t = hipblasCcopy(hcublas, n*n, d_Ao_s, 1, d_Self, 1);
   cusolverStatus = hipsolverDnCgetrf(hcusolver, n, n, d_Self, n, d_work, d_pivot, d_info);
   cusolverStatus = hipsolverDnCgetrs(hcusolver, HIPBLAS_OP_N, n, n, d_Self, n, d_pivot, d_Go, n, d_info);

   //cublasStatus = cublasCcopy(hcublas, n*n, d_Go, 1, d_Go_out, 1);

   cudaStatus = hipMemcpy(phGo_out, d_Go, n*n*sizeof(hipComplex), hipMemcpyDeviceToHost);

   cudaStatus = hipFree(d_Ao);
   cudaStatus = hipFree(d_Bo);
   cudaStatus = hipFree(d_Co);
   cudaStatus = hipFree(d_Go);
   cudaStatus = hipFree(d_Ao_s);
   cudaStatus = hipFree(d_C1);
   cudaStatus = hipFree(d_pivot);
   cudaStatus = hipFree(d_T);
   cudaStatus = hipFree(d_Self);
   cudaStatus = hipFree(d_info);
   cudaStatus = hipFree(d_work);

   return cudaStatus;

}

extern "C" int cu_Zdecimation(hipblasHandle_t hcublas, hipsolverHandle_t hcusolver, void *h_Go_out, void *h_Ao_in, void *h_Bo_in, 
       void *h_Co_in, int n, int tf32, int* ncyc, hipDoubleComplex *one, hipDoubleComplex *mone, hipDoubleComplex *zero, double SGFACC)
{
   hipError_t cudaStatus;
   hipsolverStatus_t cusolverStatus;
   hipblasStatus_t hipblasStatus_t;

   double summ;
   bool okCo = false;
   hipDoubleComplex *phGo_out = (hipDoubleComplex *) h_Go_out;
   hipDoubleComplex *phAo_in = (hipDoubleComplex *) h_Ao_in;
   hipDoubleComplex *phBo_in = (hipDoubleComplex *) h_Bo_in;
   hipDoubleComplex *phCo_in = (hipDoubleComplex *) h_Co_in;

   hipDoubleComplex *d_Ao, *d_Bo, *d_Co, *d_Go, *d_Ao_s, *d_C1, *d_T, *d_Self, *d_work;
   int *d_pivot , *d_info , Lwork ; // pivots , info , worksp . size
   int i1, NumBlocks;

   NumBlocks = ((n*n)/BLOCK_SIZE)+1;

   cudaStatus = hipMalloc(( void **)& d_Ao, n*n*sizeof(hipDoubleComplex));
   cudaStatus = hipMalloc(( void **)& d_Bo, n*n*sizeof(hipDoubleComplex));
   cudaStatus = hipMalloc(( void **)& d_Co, n*n*sizeof(hipDoubleComplex));
   cudaStatus = hipMemcpy(d_Ao, phAo_in, n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
   cudaStatus = hipMemcpy(d_Bo, phBo_in, n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
   cudaStatus = hipMemcpy(d_Co, phCo_in, n*n*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);


   hipblasStatus_t = hipblasSetPointerMode(hcublas, HIPBLAS_POINTER_MODE_HOST);
   if (tf32 == 1){
	   hipblasStatus_t = hipblasSetMathMode(hcublas, HIPBLAS_TENSOR_OP_MATH);
   }

   cudaStatus = hipMalloc(( void **)& d_Ao_s, n*n*sizeof(hipDoubleComplex));
   cudaStatus = hipMalloc(( void **)& d_C1, n*n*sizeof(hipDoubleComplex));
   cudaStatus = hipMalloc(( void **)& d_Go, n*n*sizeof(hipDoubleComplex));
   cudaStatus = hipMalloc(( void **)& d_T, n*n*sizeof(hipDoubleComplex));
   cudaStatus = hipMalloc(( void **)& d_Self, n*n*sizeof(hipDoubleComplex));
   cudaStatus = hipMalloc(( void **)& d_pivot, n*sizeof(int));
   cudaStatus = hipMalloc(( void **)& d_info, sizeof(int));

   cusolverStatus = hipsolverDnZgetrf_bufferSize(hcusolver, n, n, d_Self, n, &Lwork);
   cudaStatus = hipMalloc(( void **)& d_work, Lwork*sizeof(hipDoubleComplex));

   hipblasStatus_t = hipblasZcopy(hcublas, n*n, d_Ao, 1, d_Ao_s, 1);

   for(i1=1; i1<=300; i1++ ){
      *ncyc = i1;

      ZinitKernel<<<NumBlocks,BLOCK_SIZE>>>(d_Go, n);

      hipblasStatus_t = hipblasZcopy(hcublas, n*n, d_Ao, 1, d_Self, 1);

      cusolverStatus = hipsolverDnZgetrf(hcusolver, n, n, d_Self, n, d_work, d_pivot, d_info);
      cusolverStatus = hipsolverDnZgetrs(hcusolver, HIPBLAS_OP_N, n, n, d_Self, n, d_pivot, d_Go, n, d_info);

      hipblasStatus_t = hipblasZgemm(hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Go, n, d_Co, n, zero, d_T, n);

      hipblasStatus_t = hipblasZgemm(hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Co, n, d_T, n, zero, d_C1, n);

      hipblasStatus_t = hipblasDzasum(hcublas, n*n, d_C1, 1, &summ);

      if (summ <= SGFACC){
         if (okCo){
            break;
	 }
         else{
            okCo = true;
	 }
      }
      else{
         okCo = false;
      }

      hipblasStatus_t = hipblasZgemm(hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Bo, n, d_T, n, zero, d_Self, n);

      hipblasStatus_t = hipblasZaxpy(hcublas, n*n, mone, d_Self, 1, d_Ao_s, 1);
      hipblasStatus_t = hipblasZaxpy(hcublas, n*n, mone, d_Self, 1, d_Ao, 1);

      hipblasStatus_t = hipblasZgemm(hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Go, n, d_Bo, n, zero, d_T, n);
      hipblasStatus_t = hipblasZgemm(hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, mone, d_Co, n, d_T, n, one, d_Ao, n);

      hipblasStatus_t = hipblasZcopy(hcublas, n*n, d_C1, 1, d_Co, 1);

      hipblasStatus_t = hipblasZgemm(hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, one, d_Bo, n, d_T, n, zero, d_C1, n);

      hipblasStatus_t = hipblasZcopy(hcublas, n*n, d_C1, 1, d_Bo, 1);
   }


   ZinitKernel<<<NumBlocks,BLOCK_SIZE>>>(d_Go, n);
   hipblasStatus_t = hipblasZcopy(hcublas, n*n, d_Ao_s, 1, d_Self, 1);
   cusolverStatus = hipsolverDnZgetrf(hcusolver, n, n, d_Self, n, d_work, d_pivot, d_info);
   cusolverStatus = hipsolverDnZgetrs(hcusolver, HIPBLAS_OP_N, n, n, d_Self, n, d_pivot, d_Go, n, d_info);

   cudaStatus = hipMemcpy(phGo_out, d_Go, n*n*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

   cudaStatus = hipFree(d_info);
   cudaStatus = hipFree(d_pivot);
   cudaStatus = hipFree(d_Ao);
   cudaStatus = hipFree(d_Bo);
   cudaStatus = hipFree(d_Co);
   cudaStatus = hipFree(d_Go);
   cudaStatus = hipFree(d_Ao_s);
   cudaStatus = hipFree(d_C1);
   cudaStatus = hipFree(d_T);
   cudaStatus = hipFree(d_Self);
   cudaStatus = hipFree(d_work);

   return cudaStatus;

}

